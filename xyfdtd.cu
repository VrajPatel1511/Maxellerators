#include "hip/hip_runtime.h"
#include "xyfdtd.h"
#include <hip/hip_runtime.h>

int tag=1;
double c=2.99795e8,pi=3.14159265,eps0=8.854e-12,xmu0=1.55063706e-6,qe=1.602176487e-19,cmasse=9.10938215e-31,akb=1.3806503e-23;
double radius=0;
int rank =0, sizar=3000;
struct node *newnode(int m, int n ,int locx, int locy, int level)
{
	struct node *temp = (struct node*)malloc(sizeof(struct node));
	temp->m = m;
	temp->n = n;
	temp->locx = locx;
	temp->locy = locy;
	temp->level = level;
	return temp;
}

void mem_allocate()
{
	// ERMS = malloc(sizeof(double *) * sizar);
	// if (ERMS){
	// 	for (i = 0; i < sizar; i++){
	// 		ERMS[i] = malloc(sizeof(double) * sizar);
	// 	}
	// }

	ERMSp = (double *)malloc(sizeof(double *) * sizar);
	if (ERMSp){
		for (i = 0; i < sizar; i++){
			ERMSp[i] = malloc(sizeof(double) * sizar);
		}
	}
	erms2 = malloc(sizeof(double *) * sizar);
	if (erms2){
		for (i = 0; i < sizar; i++){
			erms2[i] = malloc(sizeof(double) * sizar);
		}
	}
	// den = malloc(sizeof(double *) * sizar);
	// if (den){
	// 	for (i = 0; i < sizar; i++){
	// 		den[i] = malloc(sizeof(double) * sizar);
	// 	}
	// }
	
	exi = malloc(sizeof(double *) * sizar);
	if (exi){
		for (i = 0; i < sizar; i++){
			exi[i] = malloc(sizeof(double) * sizar);
		}
	}
	eyi = malloc(sizeof(double *) * sizar);
	if (eyi){
		for (i = 0; i < sizar; i++){
			eyi[i] = malloc(sizeof(double) * sizar);
		}
	}
	exi1 = malloc(sizeof(double *) * sizar);
	if (exi1){
		for (i = 0; i < sizar; i++){
			exi1[i] = malloc(sizeof(double) * sizar);
		}
	}
	eyi1 = malloc(sizeof(double *) * sizar);
	if (eyi1){
		for (i = 0; i < sizar; i++){
			eyi1[i] = malloc(sizeof(double) * sizar);
		}
	}
	exs = malloc(sizeof(double *) * sizar);
	if (exs){
		for (i = 0; i < sizar; i++){
			exs[i] = malloc(sizeof(double) * sizar);
		}
	}
	eys = malloc(sizeof(double *) * sizar);
	if (eys){
		for (i = 0; i < sizar; i++){
			eys[i] = malloc(sizeof(double) * sizar);
		}
	}
	hzi = malloc(sizeof(double *) * sizar);
	if (hzi){
		for (i = 0; i < sizar; i++){
			hzi[i] = malloc(sizeof(double) * sizar);
		}
	}
	vx = malloc(sizeof(double *) * sizar);
	if (vx){
		for (i = 0; i < sizar; i++){
			vx[i] = malloc(sizeof(double) * sizar);
		}
	}
	vy = malloc(sizeof(double *) * sizar);
	if (vy){
		for (i = 0; i < sizar; i++){
			vy[i] = malloc(sizeof(double) * sizar);
		}
	}
	ext = malloc(sizeof(double *) * sizar);
	if (ext){
		for (i = 0; i < sizar; i++){
			ext[i] = malloc(sizeof(double) * sizar);
		}
	}
	eyt = malloc(sizeof(double *) * sizar);
	if (eyt){
		for (i = 0; i < sizar; i++){
			eyt[i] = malloc(sizeof(double) * sizar);
		}
	}
	exs1 = malloc(sizeof(double *) * sizar);
	if (exs1){
		for (i = 0; i < sizar; i++){
			exs1[i] = malloc(sizeof(double) * sizar);
		}
	}
	eys1 = malloc(sizeof(double *) * sizar);
	if (eys1){
		for (i = 0; i < sizar; i++){
			eys1[i] = malloc(sizeof(double) * sizar);
		}
	}
	xmid = malloc(sizeof(double) * sizar);
	ymid = malloc(sizeof(double) * sizar);
	sgdx0 = malloc(sizeof(double) * sizar);
	sgdy0 = malloc(sizeof(double) * sizar);
	DINI = malloc(sizeof(double) * sizar);
	DIFFUSION = malloc(sizeof(double *) * sizar);
	if (DIFFUSION){
		for (i = 0; i < sizar; i++){
			DIFFUSION[i] = malloc(sizeof(double) * sizar);
		}
	}
	frqio = malloc(sizeof(double *) * sizar);
	if (frqio){
		for (i = 0; i < sizar; i++){
			frqio[i] = malloc(sizeof(double) * sizar);
		}
	}
	denp = malloc(sizeof(double *) * sizar);
	if (denp){
		for (i = 0; i < sizar; i++){
			denp[i] = malloc(sizeof(double) * sizar);
		}
	}
	
	imid = malloc(sizeof(int) * sizar);
	jmid = malloc(sizeof(int) * sizar);

	exs_old = malloc(sizeof(double *) * sizar);
	if (exs_old){
		for (i = 0; i < sizar; i++){
			exs_old[i] = malloc(sizeof(double) * sizar);
		}
	}
	eys_old = malloc(sizeof(double *) * sizar);
	if (eys_old){
		for (i = 0; i < sizar; i++){
			eys_old[i] = malloc(sizeof(double) * sizar);
		}
	}
	vx_old = malloc(sizeof(double *) * sizar);
	if (vx_old){
		for (i = 0; i < sizar; i++){
			vx_old[i] = malloc(sizeof(double) * sizar);
		}
	}
	vy_old = malloc(sizeof(double *) * sizar);
	if (vy_old){
		for (i = 0; i < sizar; i++){
			vy_old[i] = malloc(sizeof(double) * sizar);
		}
	}

	root_mesh_elec = malloc(sizeof(double)*sizar);
	child_mesh_elec = malloc(sizeof(double)*sizar);
	if(root_mesh_elec && child_mesh_elec)
	{
		for (i = 0; i < sizar; ++i)
		{
			root_mesh_elec[i] = calloc(sizar, sizeof(double));
			child_mesh_elec[i] = calloc(sizar, sizeof(double));	
		}
	}

	root_mesh_mag = malloc(sizeof(double)*sizar);
	child_mesh_mag = malloc(sizeof(double)*sizar);
	if(root_mesh_mag && child_mesh_mag)
	{
		for (i = 0; i < sizar; ++i)
		{
			root_mesh_mag[i] = calloc(sizar, sizeof(double));
			child_mesh_mag[i] = calloc(sizar, sizeof(double));	
		}
	}

	root_mesh_den = malloc(sizeof(double)*sizar);
	child_mesh_den = malloc(sizeof(double)*sizar);
	if(root_mesh_den && child_mesh_den)
	{
		for (i = 0; i < sizar; ++i)
		{
			root_mesh_den[i] = calloc(sizar, sizeof(double));
			child_mesh_den[i] = calloc(sizar, sizeof(double));	
		}
	}

	c_exs = malloc(sizeof(double *) * sizar);
	if (c_exs){
		for (i = 0; i < sizar; i++){
			c_exs[i] = calloc(sizar, sizeof(double));
		}
	}
	c_eys = malloc(sizeof(double *) * sizar);
	if (c_eys){
		for (i = 0; i < sizar; i++){
			c_eys[i] = calloc(sizar, sizeof(double));
		}
	}
	c_eyi = malloc(sizeof(double *) * sizar);
	if (c_eyi){
		for (i = 0; i < sizar; i++){
			c_eyi[i] = calloc(sizar, sizeof(double));
		}
	}
	c_eyi1 = malloc(sizeof(double *) * sizar);
	if (c_eyi1){
		for (i = 0; i < sizar; i++){
			c_eyi1[i] = calloc(sizar, sizeof(double));
		}
	}

	c_erms2 = malloc(sizeof(double *) * sizar);
	if (c_erms2){
		for (i = 0; i < sizar; i++){
			c_erms2[i] = malloc(sizeof(double) * sizar);
		}
	}
	// den = malloc(sizeof(double *) * sizar);
	// if (den){
	// 	for (i = 0; i < sizar; i++){
	// 		den[i] = malloc(sizeof(double) * sizar);
	// 	}
	// }
	c_vx = malloc(sizeof(double *) * sizar);
	if (c_vx){
		for (i = 0; i < sizar; i++){
			c_vx[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_vy = malloc(sizeof(double *) * sizar);
	if (c_vy){
		for (i = 0; i < sizar; i++){
			c_vy[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_exi = malloc(sizeof(double *) * sizar);
	if (c_exi){
		for (i = 0; i < sizar; i++){
			c_exi[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_exi1 = malloc(sizeof(double *) * sizar);
	if (c_exi1){
		for (i = 0; i < sizar; i++){
			c_exi1[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_ext = malloc(sizeof(double *) * sizar);
	if (c_ext){
		for (i = 0; i < sizar; i++){
			c_ext[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_eyt = malloc(sizeof(double *) * sizar);
	if (c_eyt){
		for (i = 0; i < sizar; i++){
			c_eyt[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_exs1 = malloc(sizeof(double *) * sizar);
	if (c_exs1){
		for (i = 0; i < sizar; i++){
			c_exs1[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_eys1 = malloc(sizeof(double *) * sizar);
	if (c_eys1){
		for (i = 0; i < sizar; i++){
			c_eys1[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_xmid = malloc(sizeof(double) * sizar);
	c_ymid = malloc(sizeof(double) * sizar);
	c_sgdx0 = malloc(sizeof(double) * sizar);
	c_sgdy0 = malloc(sizeof(double) * sizar);
	c_DINI = malloc(sizeof(double) * sizar);
	c_DIFFUSION = malloc(sizeof(double *) * sizar);
	if (c_DIFFUSION){
		for (i = 0; i < sizar; i++){
			c_DIFFUSION[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_frqio = malloc(sizeof(double *) * sizar);
	if (c_frqio){
		for (i = 0; i < sizar; i++){
			c_frqio[i] = malloc(sizeof(double) * sizar);
		}
	}
	c_denp = malloc(sizeof(double *) * sizar);
	if (c_denp){
		for (i = 0; i < sizar; i++){
			c_denp[i] = malloc(sizeof(double) * sizar);
		}
	}

  c_denpold = malloc(sizeof(double *) * sizar);
	if (c_denpold){
		for (i = 0; i < sizar; i++){
			c_denpold[i] = malloc(sizeof(double) * sizar);
		}
	}
  
  c_exold = malloc(sizeof(double *) * sizar);
	if (c_exold){
		for (i = 0; i < sizar; i++){
			c_exold[i] = malloc(sizeof(double) * sizar);
		}
	}
  
   c_eyold = malloc(sizeof(double *) * sizar);
	if (c_eyold){
		for (i = 0; i < sizar; i++){
			c_eyold[i] = malloc(sizeof(double) * sizar);
		}
	}
 
  c_vxold = malloc(sizeof(double *) * sizar);
	if (c_vxold){
		for (i = 0; i < sizar; i++){
			c_vxold[i] = malloc(sizeof(double) * sizar);
		}
	} 
 
  c_vyold = malloc(sizeof(double *) * sizar);
	if (c_vyold){
		for (i = 0; i < sizar; i++){
			c_vyold[i] = malloc(sizeof(double) * sizar);
		}
	}
  
  c_hzold = malloc(sizeof(double *) * sizar);
	if (c_hzold){
		for (i = 0; i < sizar; i++){
			c_hzold[i] = malloc(sizeof(double) * sizar);
		}
	}
   
	c_hzi = malloc(sizeof(double *) * sizar);
	if (c_hzi){
		for (i = 0; i < sizar; i++){
			c_hzi[i] = malloc(sizeof(double) * sizar);
		}
	}
	
	c_imid = malloc(sizeof(int) * sizar);
	c_jmid = malloc(sizeof(int) * sizar);
}

void read_and_assign()
{	
	fptr=fopen("xstart.dat","r");
	//x0b=(nx-1)*ds;
    AA1=15.0*1.0e2;
    BB1=365.0*1.0e2;
    AA2=8.8050*1.0e2;
    BB2=258.450*1.0e2;
    AA3=0.0050*1.0e2;
    BB3=200.0*1.0e2;
    EE3=3.125e3;

    char *line = NULL;
    size_t len = 0;
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    tstop=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    nlamb=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    slambx=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    slamby=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    E0=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    PRESSURE=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    FREQ=atof(line);
    len=0;


    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    IABSOR=atof(line);
    len=0;

    if(nini<=0)
    	nini=1;  
    
    K=1;
    
    for(K;K<=nini;K++)
    {
	    getline(&line, &len, fptr);
	    getline(&line, &len, fptr);
	    len=10;
	    getline(&line, &len, fptr);
	    xmid[K]=atof(line);
	    // printf("xmid[%d]=%f\n",K,xmid[K] );
	    len=0;

	    getline(&line, &len, fptr);
	    getline(&line, &len, fptr);
	    len=10;
	    getline(&line, &len, fptr);
	    ymid[K]=atof(line);
	    // printf("ymid[%d]=%f\n",K,ymid[K] );
	    len=0;

	    getline(&line, &len, fptr);
	    getline(&line, &len, fptr);
	    len=10;
	    getline(&line, &len, fptr);
	    sgdx0[K]=atof(line);
	    // printf("sgdx0[%d]=%f\n",K,sgdx0[K] );
	    len=0;

	    getline(&line, &len, fptr);
	    getline(&line, &len, fptr);
	    len=10;
	    getline(&line, &len, fptr);
	    sgdy0[K]=atof(line);
	    // printf("sgdy0[%d]=%f\n",K,sgdy0[K] );
	    len=0;

	    getline(&line, &len, fptr);
	    getline(&line, &len, fptr);
	    len=10;
	    getline(&line, &len, fptr);
	    DINI[K]=atof(line);
	    len=0; 
    }
      
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    icpling=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    nmaxwell=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    ndifmax=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    naccel=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    NECRIR=atof(line);
    len=0; 
      
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    crec=atof(line);
    len=0;

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    cene=atof(line);
    len=0; 

    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    factor=atof(line);
    len=0; 
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    xpos=atof(line);
    len=0; 
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    ypos=atof(line);
    len=0; 
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    xs=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    ys=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    xe=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    ye=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    incrx=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    incry=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    istop=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    bxsize=atof(line);
    len=0;
    
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    bysize=atof(line);
    len=0;
   
    getline(&line, &len, fptr);
    getline(&line, &len, fptr);
    len=10;
    getline(&line, &len, fptr);
    option=atof(line);
    len=0;
    
    printf("xpos=%f ypos=%f xs=%f ys=%f xe=%f ye=%f incrx=%f incry=%f istop=%f bxsize=%f bysize=%f option=%d\n",xpos,ypos,xs,ys,xe,ye,incrx,incry,istop,bxsize,bysize,option);
    fclose(fptr);
}


void ZERO()
{
    int i;
    for(i=0;i<=nx;i++)
    {   
        for(j=0;j<=ny;j++)
        {
            // ERMS[i][j]=0.0;
            ERMSp[i][j]=0.0;
            erms2[i][j]=0.0;
            // den[i][j]=0.0;
            denp[i][j]=0.0;
            exi[i][j]=0.0;
            eyi[i][j]=0.0;
            exi1[i][j]=0.0;
            eyi1[i][j]=0.0;
            exs[i][j]=0.0;
            eys[i][j]=0.0;
            hzi[i][j]=0.0;
            vx[i][j]=0.0;
            vy[i][j]=0.0;
            ext[i][j]=0.0;
            eyt[i][j]=0.0;
            exs1[i][j]=0.0;
            eys1[i][j]=0.0;
            frqio[i][j]=0.0;
            DIFFUSION[i][j]=0.0;
        }
    }
    PARC=0.0;

    for(i=0;i<=nx*factor;i++)
    {   
        for(j=0;j<=ny*factor;j++)
        {	
        	// printf("%d %d\n", i,j);
		    c_erms2[i][j]=0.0;
            // den[i][j]=0.0;
            c_denp[i][j]=0.0;
            c_denpold[i][j]=0.0;
            c_exold[i][j]=0.0;
            c_eyold[i][j]=0.0;
            c_vxold[i][j]=0.0;
            c_vyold[i][j]=0.0;
            c_hzold[i][j]=0.0;
            c_eyi[i][j]=0.0;            
            c_eyi1[i][j]=0.0;
            c_exi[i][j]=0.0;            
            c_exi1[i][j]=0.0;
            // c_ext[i][j]=0.0
            c_exs[i][j]=0.0;
            c_eys[i][j]=0.0;
            // hzs[i][j]=0.0;
            c_vx[i][j]=0.0;
            c_vy[i][j]=0.0;
            c_ext[i][j]=0.0;
            c_eyt[i][j]=0.0;
            c_exs1[i][j]=0.0;
            c_eys1[i][j]=0.0;
            c_frqio[i][j]=0.0;
            c_DIFFUSION[i][j]=0.0;
            c_hzi[i][j]=0.0;
        }
    }
}

void free_all()
{
    // free(ERMS);
    free(ERMSp);
    free(erms2);
    // free(den);
    free(exi);
    free(eyi);
    free(exi1);
    free(eyi1);
    free(exs);
    free(eys);
    free(hzi);
    free(vx);
    free(vy);
    free(ext);
    free(eyt);
    free(exs1);
    free(eys1);
    free(xmid);
    free(ymid);
    free(sgdx0);
    free(sgdy0);
    free(DINI);
    free(DIFFUSION);
    free(frqio);
    free(denp);
    free(imid);
    free(jmid);

    free(c_erms2);
    // free(den);
    free(c_exi);
    free(c_eyi);
    free(c_exi1);
    free(c_eyi1);
    free(c_exs);
    free(c_eys);
    // free(hzs);
    free(c_vx);
    free(c_vy);
    free(c_ext);
    free(c_eyt);
    free(c_exs1);
    free(c_eys1);
    free(c_xmid);
    free(c_ymid);
    free(c_sgdx0);
    free(c_sgdy0);
    free(c_DINI);
    free(c_DIFFUSION);
    free(c_frqio);
    free(c_denp);
    free(c_denpold);
    free(c_exold);
    free(c_eyold);
    free(c_vxold);
    free(c_vyold);
    free(c_hzold);
    free(c_imid);
    free(c_jmid);
    free(c_hzi);
}

void anim()
{
    //printf("In anim\n");
	double e_total;
	char fil[50];
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
	strcpy(fil,"anim/");
	strcat(fil, buffer);
	strcat(fil,".dat");

	file_xelec=fopen(fil,"w");
	
	for(j=0; j<root_den->n; j++)
	{
		for(i=0; i<root_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_den->mesh[i][j]));
			if (root_den->mesh[i][j]!=0)
			{
				// printf("%.18f\n",child_den->mesh[i][j]);
			}
		}
		fprintf(file_xelec,"\n");
	}
	// printf("Okay, printed\n");
	fclose(file_xelec);
}

void panim()
{
    //printf("In anim\n");
	double e_total;
	char fil[50];
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", ansetup2);
	strcpy(fil,"panim/");
	strcat(fil, buffer);
	strcat(fil,".dat");

	file_xelec=fopen(fil,"w");
	
	for(j=0; j<root_den->n; j++)
	{
		for(i=0; i<root_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_den->mesh[i][j]));
			if (root_den->mesh[i][j]!=0)
			{
				// printf("%.18f\n",child_den->mesh[i][j]);
			}
		}
		fprintf(file_xelec,"\n");
	}
	
	fclose(file_xelec);
}

void canim()
{
    //printf("In anim\n");
	double e_total;
	char fil[50];
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", ansetup2);
	strcpy(fil,"canim/");
	strcat(fil, buffer);
	strcat(fil,".dat");

	file_xelec=fopen(fil,"w");
	
	for(j=0; j<child_den->n; j++)
	{
		for(i=0; i<child_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(child_den->mesh[i][j]));
			if (root_den->mesh[i][j]!=0)
			{
				// printf("%.18f\n",child_den->mesh[i][j]);
			}
		}
		fprintf(file_xelec,"\n");
	}
	
	fclose(file_xelec);
}

void canim2()
{
    //printf("In anim\n");
	double e_total;
	char fil[50];
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", amsetup2);
	strcpy(fil,"canim2/");
	strcat(fil, buffer);
	strcat(fil,".dat");

	file_xelec=fopen(fil,"w");
	
	for(j=0; j<child_den->n; j++)
	{
		for(i=0; i<child_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(child_den->mesh[i][j]));
			if (root_den->mesh[i][j]!=0)
			{
				// printf("%.18f\n",child_den->mesh[i][j]);
			}
		}
		fprintf(file_xelec,"\n");
	}
	
	fclose(file_xelec);
}

void canim3()
{
    //printf("In anim\n");
	double e_total;
	char fil[50];
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", amsetup3);
	strcpy(fil,"canim3/");
	strcat(fil, buffer);
	strcat(fil,".dat");

	file_xelec=fopen(fil,"w");
	
	for(j=0; j<child_den->n; j++)
	{
		for(i=0; i<child_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(child_den->mesh[i][j]));
			if (root_den->mesh[i][j]!=0)
			{
				// printf("%.18f\n",child_den->mesh[i][j]);
			}
		}
		fprintf(file_xelec,"\n");
	}
	
	fclose(file_xelec);
} 

void panimE()
{
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ansetup2);
    strcpy(fil,"panimE/");
    strcat(fil, buffer);
    strcat(fil,".dat");

    file_xelec=fopen(fil,"w");
    
    for(j=0; j<root_elec->n; j++)
	{
		for(i=0; i<root_elec->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_elec->mesh[i][j]));
		}
		fprintf(file_xelec,"\n");
	}
    fclose(file_xelec);
}

void canimE()
{
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ansetup2);
    strcpy(fil,"canimE/");
    strcat(fil, buffer);
    strcat(fil,".dat");

    file_xelec=fopen(fil,"w");
    
    for(j=0; j<child_elec->n; j++)
	{
		for(i=0; i<child_elec->m; i++)
		{
			fprintf(file_xelec,"%e ",(child_elec->mesh[i][j]));
		}
		fprintf(file_xelec,"\n");
	}
    fclose(file_xelec);
}


void denytempo()
{
  	
	char fil[50];
  int num=1;
  char *nam1="denytempo1";
  char *nam2="denytempo2";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
  sprintf(fil,"%s.dat",nam1);

	file_xelec=fopen(fil,"a");
 //!j=0.74*ny,0.26*ny, i=(0.35+0.03)*nx
	i=((xpos-0.4)+0.03)*nx;
  j=(ypos+0.24)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);
  
  sprintf(fil,"%s.dat",nam2);
  file_xelec=fopen(fil,"a");
  
  i=((xpos-0.4)+0.03)*nx;
  j=(ypos-0.24)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);
  
}

void Eytempo()
{
	  char fil[50];
    int num=1;
    char *nam1="Eytempo1" ;
    char *nam2="Eytempo2" ;
    char *nam3="Eytempo3" ;
    char *nam4="Eytempo4" ;
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", num);
    sprintf(fil,"%s.dat",nam1);

    file_xelec=fopen(fil,"a");
    //!i=0.75*nx,0.55*nx,0.35*nx,0.15*nx, j=0.5*ny
    j=ypos*ny;
    i= (xpos)*nx;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),eyt[i][j]);
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam2);
    file_xelec=fopen(fil,"a");
    j=ypos*ny;
    i= (xpos-0.2)*nx;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),eyt[i][j]);
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam3);
    file_xelec=fopen(fil,"a");
    j=ypos*ny;
    i= (xpos-0.4)*nx;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),eyt[i][j]);
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam4);
    file_xelec=fopen(fil,"a");
    j=ypos*ny;
    i= (xpos-0.6)*nx;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),eyt[i][j]);
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
}

void denytempocent()
{
  	
	char fil[50];
  int num=1;
  char *nam1="denyce1";
  char *nam2="denyce2";
  char *nam3="denyce3";
  char *nam4="denyce4";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
  sprintf(fil,"%s.dat",nam1);

	file_xelec=fopen(fil,"a");
 //j=0.74*ny,0.26*ny, i=(0.35+0.03)*nx
	//i=((xpos-0.4)+0.03)*nx;
  i=((xpos))*nx;
  j=(ypos)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);
  
  sprintf(fil,"%s.dat",nam2);
  file_xelec=fopen(fil,"a");
  
  i=((xpos))*nx;
  j=(ypos+0.15)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);
  
  sprintf(fil,"%s.dat",nam3);
  file_xelec=fopen(fil,"a");
  
  i=((xpos)+0.02)*nx;
  j=(ypos+0.2)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);
  
  sprintf(fil,"%s.dat",nam4);
  file_xelec=fopen(fil,"a");
  
  i=((xpos-0.05))*nx;
  j=(ypos)*ny;
  
  fprintf(file_xelec,"%lf\t%e",(t*1.0e9),(root_den->mesh[i][j]));
  fprintf(file_xelec,"\n");
  fclose(file_xelec);

}

void Eytempocent()
{
    char fil[50];
    int num=1;
    char *nam1="Eyce1" ;
    char *nam2="Eyce2" ;
    char *nam3="Eyce3" ;
    char *nam4="Eyce4" ;
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", num);
    sprintf(fil,"%s.dat",nam1);

    file_xelec=fopen(fil,"a");
    
    //j=ypos*ny;
    //i= (xpos)*nx;
    i=((xpos))*nx;
    j=(ypos)*ny;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),fabs(eyt[i][j]));
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam2);
    file_xelec=fopen(fil,"a");
    //j=ypos*ny;
    //i= (xpos-0.2)*nx;
    i=((xpos))*nx;
    j=(ypos+0.15)*ny;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),fabs(eyt[i][j]));
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam3);
    file_xelec=fopen(fil,"a");
    //j=ypos*ny;
    //i= (xpos-0.4)*nx;
    i=((xpos)+0.02)*nx;
    j=(ypos+0.2)*ny;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),fabs(eyt[i][j]));
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
    sprintf(fil,"%s.dat",nam4);
    file_xelec=fopen(fil,"a");
    //j=ypos*ny;
    //i= (xpos-0.6)*nx;
    i=((xpos-0.05))*nx;
    j=(ypos)*ny;
    
    fprintf(file_xelec,"%lf\t%e",(t*1.0e9),fabs(eyt[i][j]));
    fprintf(file_xelec,"\n");
    fclose(file_xelec);
    
}

void denx()
{
    //printf("In anim\n");
	
	char fil[50];
  int num=1;
  char *nam="denx";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
	sprintf(fil,"%s.dat",nam);
  
	file_xelec=fopen(fil,"a");
 
	j=ypos*ny;
	
		for(i=0; i<root_den->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_den->mesh[i][j]));
			
		}
			fprintf(file_xelec,"\n");
	
	fclose(file_xelec);
}
void deny()
{
    
	
	char fil[50];
  int num=1;
  char *nam="deny";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
  sprintf(fil,"%s.dat",nam);

	file_xelec=fopen(fil,"a");
 
	i=(xpos*nx)+4;
	for(j=0; j<root_den->n; j++)
	{
		
			fprintf(file_xelec,"%e ",(root_den->mesh[i][j]));
			
	}
		fprintf(file_xelec,"\n");
	
	
	fclose(file_xelec);
}
void diffux()
{
    
	char fil[50];
  int num=1;
  char *nam="diffux";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
	sprintf(fil,"%s.dat",nam);

	file_xelec=fopen(fil,"a");
	j=ypos*ny;
 
		for(i=0; i<root_den->m; i++)
		{
			fprintf(file_xelec,"%f ",(DIFFUSION[i][j]));
			
		}
		fprintf(file_xelec,"\n");
	
	fclose(file_xelec);
}
void diffuy()
{
    
	char fil[50];
  int num=1;
  char *nam="diffuy";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
	sprintf(fil,"%s.dat",nam);

	file_xelec=fopen(fil,"a");
   i=(xpos*nx)+4;
	
	for(j=0; j<root_den->n; j++)
	{
		
			fprintf(file_xelec,"%f ",(DIFFUSION[i][j]));
		
	}
		fprintf(file_xelec,"\n");
	
	fclose(file_xelec);
}
void ionfreqx()
{
    
	char fil[50];
  int num=1;
  char *nam="ionfrx";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
	sprintf(fil,"%s.dat",nam);

	file_xelec=fopen(fil,"a");
	j=ypos*ny;
		for(i=0; i<root_den->m; i++)
		{
			fprintf(file_xelec,"%e ",((root_den->mesh[i][j])*frqio[i][j]));
			
		}
		fprintf(file_xelec,"\n");
	
	fclose(file_xelec);
}
void ionfreqy()
{
    
	char fil[50];
  int num=1;
  char *nam="ionfry";
	char* buffer = (char *)malloc(sizeof(int));
	snprintf(buffer, sizeof(buffer) - 1, "%d", num);
	sprintf(fil,"%s.dat",nam);

	file_xelec=fopen(fil,"a");
	i=(xpos*nx)+4;
 
	for(j=0; j<root_den->n; j++)
	{
	
			fprintf(file_xelec,"%e ",((root_den->mesh[i][j])*frqio[i][j]));
			
	}
		fprintf(file_xelec,"\n");
	
	fclose(file_xelec);
}
void animE()
{
	  char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
    strcpy(fil,"animE/");
    strcat(fil, buffer);
    strcat(fil,".dat");

    file_xelec=fopen(fil,"w");
    
    for(j=0; j<root_elec->n; j++)
	{
		for(i=0; i<root_elec->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_elec->mesh[i][j]));
		}
		fprintf(file_xelec,"\n");
	}
    fclose(file_xelec);
}
void Ermsx()
{
	  char fil[50];
    int num=1;
    char *nam="Ermsx" ;
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", num);
    sprintf(fil,"%s.dat",nam);

    file_xelec=fopen(fil,"a");
    j=ypos*ny;
    
		for(i=0; i<root_elec->m; i++)
		{
			fprintf(file_xelec,"%e ",(root_elec->mesh[i][j]));
		}
		fprintf(file_xelec,"\n");
	
    fclose(file_xelec);
}
void Ermsy()
{
	  char fil[50];
    int num=1;
    char *nam="Ermsy";
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", num);
    sprintf(fil,"%s.dat",nam);

    file_xelec=fopen(fil,"a");
    i=(xpos*nx)+4;
    
    for(j=0; j<root_elec->n; j++)
	{
		
			fprintf(file_xelec,"%e ",(root_elec->mesh[i][j]));
	}
		fprintf(file_xelec,"\n");
	
    fclose(file_xelec);
}

void bothdenEx()
{
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
    sprintf(fil,"bothdenEx/denE%d.txt",ani);

    file_xelec=fopen(fil,"w");
  	j=ypos*ny; 
		
	      for(i=0; i<root_elec->m; i++)
		{
			fprintf(file_xelec,"%d %d %e %e \n",i,j,(root_den->mesh[i][j]),(root_elec->mesh[i][j]));
		}
    fclose(file_xelec);
}

void bothdenEy()
{
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
    sprintf(fil,"bothdenEy/denE%d.txt",ani);

    file_xelec=fopen(fil,"w");
  	i=(xpos*nx)+4;
		
	       for(j=1; j<root_elec->n; j++)
		{
		 

			fprintf(file_xelec,"%d %d %e %e \n",j,i,(root_den->mesh[i][j]),(root_elec->mesh[i][j]));
		 } 
    fclose(file_xelec);
}

void parentgrid()
{
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
    sprintf(fil,"parentgrid/par%d.txt",ani);
   // strcpy(fil,"bothdenE/");
  //  strcat(fil, buffer);
  //  strcat(fil,".dat");

    file_xelec=fopen(fil,"w");
  	//i=(1*nx/4)+4; 
   fprintf(file_xelec,"i j locx locy m n\n");
  for(j=0; j<=root_elec->n; j++)
	{
		for(i=0; i<=root_elec->m; i++)
		{
			fprintf(file_xelec,"%d %d %d %d %d %d\n",i,j,(root_elec->locx),(root_elec->locy),(root_elec->m),(root_elec->n));
		}
		fprintf(file_xelec,"\n");
	}
    fclose(file_xelec);
}

void childgrid()
{
  //int factor=1;
	char fil[50];
    char* buffer = (char *)malloc(sizeof(int));
    snprintf(buffer, sizeof(buffer) - 1, "%d", ani);
    sprintf(fil,"childgrid/chi%d.txt",ani);
   // strcpy(fil,"bothdenE/");
  //  strcat(fil, buffer);
  //  strcat(fil,".dat");

    file_xelec=fopen(fil,"w");
  //	i=(1*nx/4)+4; 
  fprintf(file_xelec,"i j xlen ylen locx locy m n\n");
   //for(j=0; j<=(root_elec->children[0])->n; j++)
    for(j=0; j<=child_elec->n; j+=1)
	{
		for(i=0; i<=child_elec->m; i+=1)
		{
		        
     // fprintf(file_xelec,"%d %d %f %f %d %d %d %d\n",i,j,(root_elec->children[0])->locx+i/4.0,(root_elec->children[0])->locy+j/4.0,(root_elec->children[0])->locx,(root_elec->children[0])->locy,(root_elec->children[0])->m, (root_elec->children[0])->n);
     fprintf(file_xelec,"%d %d %d %d %d %d %d %d\n",i,j, (child_elec->locx)*factor+i, (child_elec->locy)*factor+j, child_elec->locx, child_elec->locy, child_elec->m, child_elec->n);
		}
		fprintf(file_xelec,"\n");
	}
    fclose(file_xelec);
}
