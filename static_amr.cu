#include "hip/hip_runtime.h"
//! the following C-code numerically solves the Maxwell-Plasma continuity equation
//! to model the HPM air/gas breakdown and induced plasma and its interaction with the EM wave (high power HF microwave)
//! The code provides analysis of the phenomenon in terms of the plasmoid-->streamer growth --> discrete filaments 
//! propagates towards the Microwave source.
//! The study reduces the huge simulation time by implementing a staic Mesh Refinement (MR) and further a dynamic MR technique 
//! as it's extension.
//! various subroutines are used for calcuting the Maxwells and Plasma continuity equation both for the coarse and Fine mesh 

//! sequence of files that are used in the building the executable file static_amr 
//! main file static_amr.c, the inputs (the dynamic arrays, the inputs are read from file xstart.dat as well as constants) 
//! in xyfdtd.c
//! The postproceesing subroutines are written in the xyfdtd.c file.
//! All the global variables and arrays are declared in the xyfdtd.h . 
//! the parent mesh or coarse mesh data structures represented without c_ or child_ (fine mesh) 


#include "xyfdtd.h"
void SETUP(); //! has been commented in the code 
//Not used
void SETUP2();  //! currently Activated: defines the initial E-fields, the Gaussian plasma density. 
                //! Defines the initial small Mesh Refined region (Coarse and Fine). Next inside the time iteration, the Mesh expansion 
                //! uses this subroutine when certain threshold condition is satisfied (dynamic MR).
//naive implementation done
void HFIELD();  //! updates the Parent (Coarse Mesh) Magnetic field
//naive implementation done
void RMS(int k);    //! The RMS (root mean squared E-field is calculated) (Coarse mesh)
//naive implementation done
void MR_MUR(int row);   //! to implement the boundary conditions
//loop carried dependency so not possible
double FIONIZ(double EE,double PR); //! the ionization subroutine
//no loop is present
void ELEC_DENS();       //! the Electron density update subroutine (Coarse mesh)
//too many if conditions ay lead to thread divergence
void EFIELD();          //! the Electric field update subroutine   (Coarse mesh)
//boundary conition and too many variables
void child_HFIELD();    //! the child magnetic field subroutine
//naive implementation done
void child_RMS(int k);  //! child RMS E-field is calcultated here
//naive implementation done
//! avoid the following child MUR subroutines only parent MUR boundary is required for the scattered wave absorption
void child_MR_MUR_0(int row);  
void child_MR_MUR_1(int row);
void child_MR_MUR_2(int row);
void child_MR_MUR_3(int row);
//------------------------------
void child_ELEC_DENS();         //! subroutine calculates child electron density
//same as ELEC_DENS
void child_EFIELD();            //! subroutine calculates child electric field
//same as EFIELD()
//! subroutines to interpolate the Parent(Coarse mesh) data for E-,H-,Electron density to child (fine mesh)
void interpolatex(double a);   // ! along x-direction row wise ( only on the Parent-child boundary)
void interpolatey(double a);    //! along y-direction columnwise    ( only on the Parent-child boundary)
void interpolatecorners(double a); //! along the corners 
void interpolatecornersnew(double a); //! avoid
void interpolatexall(double a);     //! avoid
void interpolatexinitial(double a); //! for obtaining the initial parent gaussian density profile to child mesh
void interpolatexnew(double a);     //! interpolate all the initial parent data to child along x expansion of box (mesh)
void interpolateynew(double a);     //! interpolate all the initial parent data to child along y expansion of box (mesh)     
void interpolateyall(double a);     //! interpolate all the initial parent data to child along y (parent gaussian density)
void c2p();                         //! to transfer the child (fine mesh ) data to parent mesh

//! declare all the time variables (structure type) as well as double variables to store times 
struct timeval begin, end, total_start, total_end,program_start,program_end,t_panimstar,t_panimend,telcend;

// varaibled for noting time of Calculations for each subroutine 
double t_cal_hfield,t_cal_efield,t_cal_rms,t_cal_elec_dens,t_cal_anim,t_cal_c2p,t_cal_interpolatex,t_cal_interpolatexinitial,t_cal_interpolatexnew,t_cal_interpolateynew;
double t_cal_child_hfield,t_cal_child_efield,t_cal_child_rms,t_cal_child_elec_dens;

double t_panimcal;
double t1, t_efield_hfield,t_elec_dens,t_anim,t_rms,t_zero,t_vel_x,t_vel_y;
//! declare all the file pointers used/to be used in the code  
FILE *fptr3,*fx1,*fx2,*fx3,*fy1,*fy2,*fy3,*fxt,*frefine,*canitimxy,*velp1,*velp2,*velp3;

__global__ void HFIELD(struct node * grid,double ** exs,double ** eys, double *dtmds)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
 int j = blockIdx.y*blockDim.y+threadIdx.y;
 
   
    if ( i < grid->m && j<grid->n )
    {
        grid->mesh[i][j]+= (-(eys[i+1][j]-eys[i][j])+(exs[i][j+1]-exs[i][j]))*(*dtmds);
    }
}

__global__ void RMS(struct node * root_elec,double z1,double z2,double *inv_nperdt,double **ext,double **eyt,double **ERMSp,double **erms2,int *k)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    if ( i < root_elec->m && j<root_elec->n )
    {
        z1=(ext[i][j]*ext[i][j]+ext[i-1][j]*ext[i-1][j])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
        z2=(eyt[i][j]*eyt[i][j]+eyt[i][j-1]*eyt[i][j-1])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
        ERMSp[i][j] = erms2[i][j];
        erms2[i][j]=erms2[i][j]+(z1+z2)*(*inv_nperdt);     //! time updates and averages (parent)
        if(*k==2)
        {
            if (erms2[i][j]<0)
            	{
            		printf("Alert!!\n");
            	}
                root_elec->mesh[i][j] = sqrt(erms2[i][j]);  //! completes a period and then squre root the time avg data (parent)
                erms2[i][j]=0.0f;
        }
    }
}

__global__ void child_RMS(struct node * root_elec,double z1,double z2,double *inv_nperdt,double **ext,double **eyt,double **erms2,int *k)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    if ( i < root_elec->m && j<root_elec->n )
    {
        z1=(ext[i][j]*ext[i][j]+ext[i-1][j]*ext[i-1][j])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
        z2=(eyt[i][j]*eyt[i][j]+eyt[i][j-1]*eyt[i][j-1])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
        erms2[i][j]=erms2[i][j]+(z1+z2)*(*inv_nperdt);     //! time updates and averages (parent)
        if(*k==2)
        {
            if (erms2[i][j]<0)
            	{
            		printf("Alert!!\n");
            	}
                root_elec->mesh[i][j] = sqrt(erms2[i][j]);  //! completes a period and then squre root the time avg data (parent)
                erms2[i][j]=0.0f;
        }
    }
}

__global__ void setup_init(struct node *dev_root_elec,struct node * dev_den,double *E0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; 
    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    if(i<dev_root_elec->m && j>dev_root_elec->n)
    {
        dev_den->mesh[i][j] = 0.0;
  		dev_root_elec->mesh[i][j] = (*E0)/sqrt(2.0);
    }
}

__global__ void setup_init1(struct node * root_den,int *ny,int *nx,double *xxi,double *ds,double *ardix,double *yyj,double *ardiy,double *xd0,double *yd0,double *dinig,double * sgdx0,double *sgdy0,double *DINI, int *K)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; 
    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    if(i<*nx && j<*ny)
    {
        *xxi=(*ds)*i;
	    *ardix=0.0;
	    if(sgdx0[1]>0)
	        *ardix=(-pow(((*xxi)-(*xd0)),2))/2.0/sgdx0[1]/sgdx0[1];
        *yyj=(*ds)*j;
        *ardiy=0.0;
        if(sgdy0[1]>0) 
            *ardiy=-pow((*(yyj)-(*yd0)),2)/2.0/sgdy0[*K]/sgdy0[*K];
            *dinig=DINI[*K]*exp((*ardix)+(*ardiy));
            if(*dinig<=1.0e13)
                *dinig=0;
                 
        root_den->mesh[i][j] = root_den->mesh[i][j]+ (*dinig);
    }
}

int main()
{

  gettimeofday(&program_start, NULL);
	mem_allocate();        //! inside the xyfdtd.c file : all the dynamic arrays are declared 
	read_and_assign();     //! inside the xyfdtd.c file: all the inputs are read from the xstart.dat file and assigned to corresponding global varibles 
   
    
    //!------------------new  ( the portion of code to note the start time )
  time_t tkl; 
  struct tm *info;  
  char buffer10[64];
  
    gettimeofday(&begin, NULL);
    t1 = begin.tv_usec;
    gettimeofday(&total_start, NULL);
    tkl=total_start.tv_sec; //!new
    info = localtime(&tkl);//!new
    //!new
  printf("%s",asctime (info));
  strftime (buffer10, sizeof buffer10, "Today is %A, %B %d.\n", info);
  printf("%s",buffer10);
  strftime (buffer10, sizeof buffer10, "The time is %I:%M %p.\n", info);
  printf("%s",buffer10);
  //!----------------------new ( also print the starting time, the start time: for runtime of complete simulation and intermediates)
    nini=1;
    

    gettimeofday(&end, NULL);
    printf("Reading Input time: %f s\n", ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0)));

    //======================Caculating Constants=====================

	gettimeofday(&begin, NULL);
	t1 = begin.tv_usec;

  //! Grid sizes along x and y
	nx=nlamb*slambx;     
	ny=nlamb*slamby;
  //! peiod / number of steps to complete a period
	nperdt=2.0*nlamb;
  //! for requirement in calculations as constants
	inv_nperdt=1.0f/(double)nperdt;
	inv_c=1.0f/c;
	inv_cmasse=1.0f/cmasse;
	//!--------------
  t=0;
	ani=0;         //! not required now
  ansetup2=0;   //! only required in the code for printing simulation time for parent density plot
  amsetup2=0;   //! avoid
  amsetup3=0;   //! avoid
	//nstep=0;
  n=0;    //! number of iterations (temporal) 
	printf("nx %d, ny %d, nlamb %d, slambx %f, slamby %f\n",nx,ny,nlamb,slambx, slamby );

    //!=============================================================

    printf("Here\n");
    gettimeofday(&begin, NULL);
    ZERO();       //! initialize all the arrays to zeros 
   // SETUP();
    SETUP2();     //! to create the initial Mesh refined region, the initial gaussian density and the fileds are defined
                  //! inside do loop it is used to expand the mesh refined region dynamically (expand box)
    gettimeofday(&end, NULL);
    t_zero += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));

    gettimeofday(&end, NULL);

    printf("Initialization time: %f s\n", ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0)));


      ELEC_DENS();       //! the electron density update on parent 

    child_ELEC_DENS();  //! the electron density update on child
    
    printf("computing up to time  %e \n",tstop);

    //!=============================================================
    fptr=fopen("output.txt","w");      

    //--------------- Initial parameters ouput----------------------

    fprintf(fptr,"nx=%d   ny=%d\n",nx,ny);
    fprintf(fptr,"DS=%f\tC_DS=%f\t DT=%f\tC_DT=%f\n",ds,c_ds,dt*1.0e15,c_dt*1.0e15);
    fprintf(fptr,"Freq=%f   Omega=%f\n",FREQ,OMEG);
    fprintf(fptr,"Time period=%f\n",1.0/FREQ);
    fprintf(fptr,"Lambda=%f\n",3.0e8/FREQ);
       printf("fprintfs\n");
    //!put 0    
    fprintf(fptr,"Collision Freq=%f\n",FNUM);
    fprintf(fptr,"Recombination Coef=%f\n",RECOMB);
    fprintf(fptr,"Mobility=%f\n",EMOB);
    fprintf(fptr,"Electron Temp= (eV)%f\n",ETEM);
    fprintf(fptr,"DIFFUSION Coef=%f\n",EDIF);
    fprintf(fptr,"Initial Gas/Neutral density=%f\n",DENG0);
    fprintf(fptr,"Cutoff-density=%f\n",(eps0*cmasse/pow(qe,2))*(pow(OMEG,2)+pow(FNUM,2)));// check formula
    fprintf(fptr, "Tstop %.10lf\n",tstop); 
    fclose(fptr);
    printf("second bunch of fprintfs\n");
    
    //!=============================================================
    KELEC=0;    //! to print the electron-density/gas-density/gas-temperature/electron-temperature data on parents at discrete periods
    c_KELEC=0;  //! to print the electron-density/gas-density/gas-temperature/electron-temperature data on child at discrete periods
    n=0;        //! parent iteration (temporal)
    c_n=0;      //! child iterations (temporal)
    int f=0;

    //!=============================================================

    printf("some fopens\n");
    //! constants/coefficients used in E-field updates in parent mesh
    qmdt=qe*inv_cmasse*dt;  
  	aa=FNUM*dt/2.0; 
  	alpha=(1.0-aa)/(1.0+aa);
  	gamma1=1+aa;
  	const3=.50*qe*qe*inv_cmasse/eps0;
  	const4=dt*dt/4.0/gamma1;
  	const7=.25*dte*(1.0+alpha);
  	const8=1.0/2.0/gamma1;
  	//!-------------------
    //! for forward wave on parent (root)
    i0=2;
  	x0=(i0-1)*ds;
    //! for backward wave on parent (root)
    i01=nx;
    x01=(i01-1)*ds;
	  //!-------------------

	  //! constants/coefficients used in E-field updates in child mesh
    c_qmdt=qe*inv_cmasse*c_dt;
    c_aa=FNUM*c_dt/2.0; 
    c_alpha=(1.0-c_aa)/(1.0+c_aa);
    c_gamma1=1+c_aa;
    c_const4=c_dt*c_dt/4.0/c_gamma1;	
    c_const7=.25*c_dte*(1.0+c_alpha);
    c_const8=1.0/2.0/c_gamma1;
    //!-----------------------
    c_x0 = c_ds;                //! forward wave on child
    c_x01= (i01-1)*factor*c_ds; //! backward wave on child
    c_nperdt = nperdt*dt/c_dt;
    c_inv_nperdt = 1.0f/(double)c_nperdt;
    
    
    int icent,jcent,iloc1,iloc2,iloc3,iloc4,jloc1,jloc2,jloc3,k,iend,p1,ix,py1,py2,py3,py4,py5,py6,py7,py8;
    int count_cen,countx1,countx2,countx3,county1,county2,county3,county4,n_ini,lup,countrank,count_run,count_cen1;
    double velx1,velx2,velx3,vely1,vely2,vely3,tx1,tx2,tx3,ty1,ty2,ty3,lambda,tstart,t0,tnew;
    inter_value = 1.0/factor;   //! decide fraction for interpolated data
    
    icent=(int)(xpos*nx); //the x-center (interms of cell number)
    //! avoid for now 
    iloc1=(icent-(0.15*nx));iloc2=(icent-(0.25*nx));iloc3=(icent-(0.35*nx)); iloc4=(icent-(0.55*nx));
    
        
    //!-- ycenter (interms of cell number)
    jcent=(int)(ypos*ny);
    //! avoid for now
    jloc1=(jcent+(0.15*ny));jloc2=(jcent+(0.25*ny));jloc3=(jcent+(0.35*ny));
    //!-------------------
   
    p1=(int)(0.2*nlamb);
    //!--- location on y to detect streamer growth and expand box (mesh) along y----
    //! 10% top/bottom, 30% top/bottom, 40% top/bottom, 50% top/bottom,
    py1=jcent+(int)(0.1*jcent);
    py2=jcent-(int)(0.1*jcent);
    py3=jcent+(int)(0.3*jcent);
    py4=jcent-(int)(0.3*jcent); 
    py5=jcent+(int)(0.4*jcent);
    py6=jcent-(int)(0.4*jcent);
    py7=jcent+(int)(0.5*jcent);
    py8=jcent-(int)(0.5*jcent); 
   
   
    iend=(int)(istop*nx);   //!same as xs*nx (max mesh region extent towards source on left)
    lambda=c/FREQ;tstart=0; tnew=0;
    k=1;
    

    //!avoid now
    count_cen=-1;countx1=-1;countx2=-1;countx3=-1;
    //! to detect the streamer crossing a point on the line once to dynamically expand the mesh  
    county1=-1;county2=-1;county3=-1;county4=-1;
    //! avoid
    count_run=-1;count_cen1=-1;
 
    printf("icent=%d\tjcent=%d\tiloc1=%d\tiloc2=%d\tiloc3=%d\tjloc1=%djloc2=%d\tjloc3=%d\n",icent,jcent,iloc1,iloc2,iloc3,jloc1,jloc2,jloc3);
    printf("iend=%d \n",iend);
    
    //! print the initial mesh informations
    frefine=fopen("refregionxy.txt","a");
      fprintf(frefine,"time ");
      fprintf(frefine,"KELEC ");
      fprintf(frefine,"Tcellsrefx ");
      fprintf(frefine,"Tcellsrefy ");
      fprintf(frefine,"xstar ");
      fprintf(frefine,"xend ");
      fprintf(frefine,"ystar ");
      fprintf(frefine,"yend \n");
      fclose(frefine);
      
      frefine=fopen("refregionxy.txt","a");
      fprintf(frefine,"%lf ",t*1.0e9);
      fprintf(frefine,"%d ",KELEC);
      fprintf(frefine,"%d ",(xend-xstar)*factor);
      fprintf(frefine,"%d ",(yend-ystar)*factor);
      fprintf(frefine,"%d ",xstar);
      fprintf(frefine,"%d ",xend);
      fprintf(frefine,"%d ",ystar);
      fprintf(frefine,"%d \n",yend);
      fclose(frefine);
    //!--------------------------
      //! print the simulation and corresponding runtime for expanding mesh
      canitimxy=fopen("canitimxy.txt","a");
      fprintf(canitimxy,"KELEC ");
      fprintf(canitimxy,"sim.time(ns) ");
      fprintf(canitimxy,"run time(s) \n");
      fclose(canitimxy);

    /*    //! activate to print the front location for calculating front velocity
          velp1=fopen("velplot1.txt","a");
        	fprintf(velp1,"i ");  
          fprintf(velp1,"distance(lamb) ");
          fprintf(velp1,"time(ns) "); 
          fprintf(velp1,"density\n"); 
          fclose(velp1);
    */
    do
    {
        n=n+1;
                
        
      
     if(option==1)  //! activates only when option is 1 in xstart.dat ( for dynamic MR /AMR) else for only MR it is deactivated
     {  
       //!================= Increament Mesh size (dynamic MR) along x and y ======================
       if((root_den->mesh[xstar][jcent])>1.0e16) //! checks the density at the xstart of initial box exceeding threshold to expand along x
        {   
            chc=1;
            if(xstar<=iend)         //! stops growing as the xstart reaches equal or less than the iend=istop*nx (or xs*nx)
            {
              printf(" fine mesh has reached the boundary\n");
              break;
            }
            else{
                 
            printf("Entered SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
            SETUP2();           //! for creating expanding mesh from initial mesh along x-direction
            
           printf("leave SETUP2\n");
           printf("KELEC=%d\n",(int)(n/nperdt));
           
            frefine=fopen("refregionxy.txt","a");
            fprintf(frefine,"%lf ",t*1.0e9);
            fprintf(frefine,"%d ",KELEC);
            fprintf(frefine,"%d ",(xend-xstar)*factor);
            fprintf(frefine,"%d ",(yend-ystar)*factor);
            fprintf(frefine,"%d ",xstar);
            fprintf(frefine,"%d ",xend);
            fprintf(frefine,"%d ",ystar);
            fprintf(frefine,"%d \n",yend);
            fclose(frefine);
           }
        }
        
        for (ix=0;ix<root_den->m;ix++)
        {
            //! first threshold line (10% up/ down jcent (or ycent))
         if((root_den->mesh[ix][py1])>1.0e16 || (root_den->mesh[ix][py2])>1.0e16) //! density check at both ystart/yend of initial box exceeding threshold to expand along y
         {  
           county1+=1;
           if(county1==0){ 
            chc=2;
            
            if(ystar<=(ys*ny)|| yend>=(ye*ny) )   //! stops growing as the ystart/yend reaches equal or less than the (ys*ny)/ greater than (ye*ny)
            {
                printf(" fine mesh reached the boundary\n");
             // break;
            }
            else{
                 
            printf("Entered SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
            SETUP2();          //! for creating expanding mesh from initial mesh along y-direction
                        
            printf("leave SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
           frefine=fopen("refregionxy.txt","a");
            fprintf(frefine,"%lf ",t*1.0e9);
            fprintf(frefine,"%d ",KELEC);
            fprintf(frefine,"%d ",(xend-xstar)*factor);
            fprintf(frefine,"%d ",(yend-ystar)*factor);
            fprintf(frefine,"%d ",xstar);
            fprintf(frefine,"%d ",xend);
            fprintf(frefine,"%d ",ystar);
            fprintf(frefine,"%d \n",yend);
            fclose(frefine);
            }
           }
         }
        }
        
        for (ix=0;ix<root_den->m;ix++)
        {
          //! first threshold line (30% up/ down jcent (or ycent))
         if((root_den->mesh[ix][py3])>1.0e16 || (root_den->mesh[ix][py4])>1.0e16) //! density check at both ystart/yend of initial box exceeding threshold to expand along y
         {  
           county2+=1;
           if(county2==0){ 
             chc=2;
            
            if(ystar<=(ys*ny)|| yend>=(ye*ny) ) //! stops growing as the ystart/yend reaches equal or less than the (ys*ny)/ greater than (ye*ny)
            {
                printf(" fine mesh reached the boundary\n");
             // break;
            }
            else{
                 
            printf("Entered SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
            SETUP2();                    //! for creating expanding mesh from initial mesh along y-direction
                        
            printf("leave SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
           frefine=fopen("refregionxy.txt","a");
            fprintf(frefine,"%lf ",t*1.0e9);
            fprintf(frefine,"%d ",KELEC);
            fprintf(frefine,"%d ",(xend-xstar)*factor);
            fprintf(frefine,"%d ",(yend-ystar)*factor);
            fprintf(frefine,"%d ",xstar);
            fprintf(frefine,"%d ",xend);
            fprintf(frefine,"%d ",ystar);
            fprintf(frefine,"%d \n",yend);
            fclose(frefine);
            }
           }
         }
        }
        
         for (ix=0;ix<root_den->m;ix++)
        {
          //! first threshold line (40% up/ down jcent (or ycent))
         if((root_den->mesh[ix][py5])>1.0e16 || (root_den->mesh[ix][py6])>1.0e16)  //! density check at both ystart/yend of initial box exceeding threshold to expand along y   
         {   
            county3+=1;
           if(county3==0){ 
            chc=2;
            
            if(ystar<=(ys*ny)|| yend>=(ye*ny) )   //! stops growing as the ystart/yend reaches equal or less than the (ys*ny)/ greater than (ye*ny)
            {
                printf(" fine mesh reached the boundary\n");
             // break;
            }
            else{
                 
            printf("Entered SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
            SETUP2();           //! for creating expanding mesh from initial mesh along y-direction
                        
            printf("leave SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
           frefine=fopen("refregionxy.txt","a");
            fprintf(frefine,"%lf ",t*1.0e9);
            fprintf(frefine,"%d ",KELEC);
            fprintf(frefine,"%d ",(xend-xstar)*factor);
            fprintf(frefine,"%d ",(yend-ystar)*factor);
            fprintf(frefine,"%d ",xstar);
            fprintf(frefine,"%d ",xend);
            fprintf(frefine,"%d ",ystar);
            fprintf(frefine,"%d \n",yend);
            fclose(frefine);
            }
           }
         }
        }
        
         for (ix=0;ix<root_den->m;ix++)
        {
          //! first threshold line (50% up/ down jcent (or ycent))
         if((root_den->mesh[ix][py7])>1.0e16 || (root_den->mesh[ix][py8])>1.0e16) //! density check at both ystart/yend of initial box exceeding threshold to expand along y
         {   
            county4+=1;
           if(county4==0){ 
            chc=2;
            
            if(ystar<=(ys*ny)|| yend>=(ye*ny) )     //! stops growing as the ystart/yend reaches equal or less than the (ys*ny)/ greater than (ye*ny)
            {
                printf(" fine mesh reached the boundary\n");
             // break;
            }
            else{
                 
            printf("Entered SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
            SETUP2();                           //! for creating expanding mesh from initial mesh along y-direction
                      
            printf("leave SETUP2\n");
            printf("KELEC=%d\n",(int)(n/nperdt));
            
           frefine=fopen("refregionxy.txt","a");
            fprintf(frefine,"%lf ",t*1.0e9);
            fprintf(frefine,"%d ",KELEC);
            fprintf(frefine,"%d ",(xend-xstar)*factor);
            fprintf(frefine,"%d ",(yend-ystar)*factor);
            fprintf(frefine,"%d ",xstar);
            fprintf(frefine,"%d ",xend);
            fprintf(frefine,"%d ",ystar);
            fprintf(frefine,"%d \n",yend);
            fclose(frefine);
            }
           }
         }
        }
      }
      //!===================================================== 
        //!----------- velocity ----------
        /*
          velp1=fopen("velplot1.txt","a");    
      		
      			if((root_den->mesh[icent-k][jcent])>1.0e19)
            {
                         
             fprintf(velp1,"%d %f %lf %e \n",icent-k,((1.0/nlamb)*(float)k),t*1.0e9,(root_den->mesh[icent-k][jcent]));
             k+=1;
      		  }
             
          fclose(velp1);
        */
        
        //!------------------------------------
        
                
      	
        if((t>tstop)||(root_den->mesh[iend][jcent])>1.0e16) //! use to terminate the full code while simulating
        //if((t>tstop)||KELEC==10520)      //! use particular value of KELEC to stop code while experimenting               
        //if((t>tstop)||KELEC==4020)
        {  
	         
    	    printf("code has successfully ended\n");  
          gettimeofday(&total_end,NULL);   
           tkl=total_end.tv_sec; //!new
           info = localtime(&tkl);//!new
           //! gives info on the clock time when code ended
           printf("%s",asctime (info)); 
          t1 = ((total_end.tv_sec - total_start.tv_sec) + ((total_end.tv_usec - total_start.tv_usec)/1000000.0));
          break;
	     
        }
        gettimeofday(&begin, NULL);

        
        //HFIELD();   //! call H-field at n   t=0
        hipMalloc((void**)&dev_mag, sizeof(root_mag));
        hipMalloc((void**)&dev_exs, sizeof(exs));
        hipMalloc((void**)&dev_eys, sizeof(eys));
        hipMalloc((void**)&dev_dtmds, sizeof(dtmds));
        hipMemcpy(dev_mag, root_mag, sizeof(root_mag), hipMemcpyHostToDevice);
        hipMemcpy(dev_exs, exs, sizeof(exs), hipMemcpyHostToDevice);
        hipMemcpy(dev_eys, eys, sizeof(eys), hipMemcpyHostToDevice);
        hipMemcpy(dev_dtmds, &dtmds, sizeof(dtmds), hipMemcpyHostToDevice);
        HFIELD<<<(ceil(root_mag->m/32),ceil(root_mag->n/32)),(32,32)>>>(dev_mag,dev_exs,dev_eys,dev_dtmds);
        hipMemcpy(root_mag, dev_mag, sizeof(root_mag), hipMemcpyDeviceToHost);
        hipMemcpy(hzi, dev_mag->mesh, sizeof(root_mag), hipMemcpyDeviceToHost);

        hipFree(dev_mag);
        hipFree(dev_exs);
        hipFree(dev_eys);
        hipFree(dev_dtmds);

        EFIELD();   //! call E-field at n+1/2 time steps  t=1/2dt
        
        // hipMalloc((void**)&dev_root_elec, sizeof(root_elec));
        // hipMalloc((void**)&dev_x0, sizeof(double));
        // hipMalloc((void**)&dev_OMEG, sizeof(double));
        // hipMalloc((void**)&dev_newt, sizeof(double));
        // hipMalloc((void**)&dev_inv_c, sizeof(double));
        // hipMalloc((void**)&dev_c_dt, sizeof(double));
        // hipMalloc((void**)&dev_sine, sizeof(double));
        // hipMalloc((void**)&dev_sine1, sizeof(double));
        // hipMalloc((void**)&dev_x, sizeof(double));
        // hipMalloc((void**)&dev_c, sizeof(double));

        // hipMemcpy(dev_root_elec, root_elec, sizeof(root_elec), hipMemcpyHostToDevice);
        // hipMemcpy(dev_x0, x0, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_OMEG, OMEG, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_newt, newt, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_inv_c, inv_c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_inv_c, inv_c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c, c, sizeof(double), hipMemcpyHostToDevice);
        
        // EFIELD<<<dimGrid, dimBlock>>>(dev_elec,dev_x0,dev_OMEG,dev_newt,dev_inv_c,dev_c_dt,dev_sine,dev_sine1,dev_x,dev_c);
        
        // hipMemcpy(sine, dev_sine, sizeof(dev_sine), hipMemcpyDeviceToHost);
        // hipMemcpy(sine1, dev_sine1, sizeof(dev_sine1), hipMemcpyDeviceToHost);

        // hipFree(dev_elec);
        // hipFree(dev_x0);
        // hipFree(dev_OMEG);
        // hipFree(dev_newt);
        // hipFree(dev_inv_c);
        // hipFree(dev_c_dt);
        // hipFree(dev_sine);
        // hipFree(dev_sine1);
        // hipFree(dev_x);
        // hipFree(dev_c);

        gettimeofday(&end, NULL);
        t_efield_hfield += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
        // printf("Efield hfield done\n");
        t = t + dt;

		KRMS=1;
        //!=============================================================
        if(fmod(n,nperdt)==0)         //! when n is integral multiples of period
        	KRMS=2;

        gettimeofday(&begin, NULL);
        //RMS(KRMS);                     //! finally the RMS calculated as KRMS=2 on each period
        
        hipMalloc((void**)&dev_root_elec, sizeof(root_elec));
        hipMalloc((void**)&dev_z1, sizeof(double));
        hipMalloc((void**)&dev_z2, sizeof(double));
        hipMalloc((void**)&dev_inv_nperdt, sizeof(inv_nperdt));
        hipMalloc((void**)&dev_ext, sizeof(ext));
        hipMalloc((void**)&dev_eyt, sizeof(eyt));
        hipMalloc((void**)&dev_ERMSp, sizeof(ERMSp));
        hipMalloc((void**)&dev_erms2, sizeof(erms2));
        hipMalloc((void**)&dev_KRMS, sizeof(KRMS));

        hipMemcpy(dev_root_elec, root_elec, sizeof(root_elec), hipMemcpyHostToDevice);
        hipMemcpy(dev_inv_nperdt, &inv_nperdt, sizeof(inv_nperdt), hipMemcpyHostToDevice);
        hipMemcpy(dev_ext, ext, sizeof(ext), hipMemcpyHostToDevice);
        hipMemcpy(dev_eyt, eyt, sizeof(eyt), hipMemcpyHostToDevice);
        hipMemcpy(dev_ERMSp, ERMSp, sizeof(ERMSp), hipMemcpyHostToDevice);
        hipMemcpy(dev_erms2, erms2, sizeof(erms2), hipMemcpyHostToDevice);
        hipMemcpy(dev_KRMS, &KRMS, sizeof(int), hipMemcpyHostToDevice);

        RMS<<<(ceil(root_elec->m/32),ceil(root_elec->n/32)),(32,32)>>>(dev_root_elec, dev_z1,dev_z2,dev_inv_nperdt,dev_ext,dev_eyt,dev_ERMSp,dev_erms2,KRMS);
        
        hipMemcpy(ERMSp, dev_ERMSp, sizeof(dev_ERMSp), hipMemcpyDeviceToHost);
        hipMemcpy(erms2, dev_erms2, sizeof(dev_erms2), hipMemcpyDeviceToHost);
        if(KRMS==2)
        {
            hipMemcpy(root_elec->mesh, dev_root_elec->mesh, sizeof(dev_root_elec->mesh), hipMemcpyDeviceToHost);
        }
       
        hipFree(dev_root_elec);
        hipFree(dev_z1);
        hipFree(dev_z2);
        hipFree(dev_inv_nperdt);
        hipFree(dev_ext);
        hipFree(dev_eyt);
        hipFree(dev_ERMSp);
        hipFree(dev_erms2);
        hipFree(dev_KRMS);

        gettimeofday(&end, NULL);
        t_rms += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
        //!=============================================================
	     /* 
       if(n>=1)
       {
            ani++;
            denx();
            deny();
            diffux();
            diffuy();
            ionfreqx();
            ionfreqy();
            Ermsx();
            Ermsy();
       
       }*/
       
        //!%%%%%%% print the temporal profile of Ey-field (Total)  at i=0.75*nx,0.55*nx,0.35*nx,0.15*nx, j=0.5*ny for all time instants %%%%%%%%%%
       //Eytempo();
       //!%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
       //!%%%%%%% print the temporal profile of density at j=0.74*ny,0.26*ny, i=(0.38)*nx for all time instants %%%%%%%%%%
      // denytempo();
       //!%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
       
	     
        if(fmod(n,nperdt*nmaxwell)==0)
        {
            if(icpling!=0)
            { 
                gettimeofday(&begin, NULL);
                ELEC_DENS();                  //! Electron density is called on start of each period under RMS field
                gettimeofday(&telcend, NULL);
                t_elec_dens += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_sec)/1000000.0));
            }
            else
            {
                DTAC=1.0/FREQ;
                TIMD=(double)(n)*inv_nperdt*nmaxwell*DTAC;
            }
            //!=============================================================XXX
            KELEC=KELEC+1;
           if(n==1){
           parentgrid();
                childgrid();   
           }
          
            
                 gettimeofday(&begin, NULL);

   		 //if(KELEC==25||KELEC==50||KELEC==100||KELEC==200||KELEC==250||KELEC==500||KELEC==1000||KELEC==1200||KELEC==1500||KELEC==1600||KELEC==1825||KELEC==1850||KELEC==1925||KELEC==1950||KELEC==2000){
       // if(KELEC==25||KELEC==50||KELEC==100||KELEC==200||KELEC==500||KELEC==1000||KELEC==2000||KELEC==3000||KELEC==4000||KELEC==4200||KELEC==4400||KELEC==4800||KELEC==5000||KELEC==5200||KELEC==5400){
       //if(KELEC==25||KELEC==50||KELEC==100||KELEC==200||KELEC==500||KELEC==1000||KELEC==2000||KELEC==3000){
 //      if(KELEC==1||KELEC==50||KELEC==500||KELEC==1000||KELEC==2000||KELEC==3000||KELEC==4000||KELEC==4200||KELEC==4400||KELEC==4800||KELEC==5000||KELEC==5200||KELEC==5400||KELEC==6000||KELEC==6500||KELEC==7000||KELEC==7500||KELEC==8000||KELEC==8500||KELEC==9000||KELEC==9500||KELEC==10500||KELEC==11100){
      //if(KELEC==1||KELEC==50||KELEC==100||KELEC==200||KELEC==352||KELEC==400||KELEC==500||KELEC==650||KELEC==700||KELEC==800||KELEC==850||KELEC==900||KELEC==950||KELEC==1000||KELEC==1200||KELEC==1600||KELEC==1800||KELEC==2000||KELEC==2200||KELEC==2500||KELEC==2800||KELEC==3000){
      //if(KELEC==1||KELEC==50||KELEC==100||KELEC==200||KELEC==352||KELEC==400||KELEC==500||KELEC==650||KELEC==700||KELEC==800||KELEC==850||KELEC==900||KELEC==950||KELEC==1000||KELEC==1200||KELEC==1600||KELEC==1800||KELEC==2000||KELEC==2200||KELEC==2500||KELEC==2800||KELEC==3000||KELEC==3400||KELEC==3600||KELEC==3800||KELEC==4000||KELEC==4200||KELEC==4400||KELEC==4600||KELEC==4800||KELEC==5000||KELEC==5200||KELEC==5400){
      // if(KELEC==1||KELEC==50||KELEC==900||KELEC==905||KELEC==910||KELEC==915||KELEC==920||KELEC==925||KELEC==930||KELEC==935||KELEC==940||KELEC==945||KELEC==950){
      if(KELEC==1||KELEC==50||KELEC==100||KELEC==200||KELEC==352||KELEC==400||KELEC==500||KELEC==650||KELEC==700||KELEC==800||KELEC==850||KELEC==900||KELEC==950||KELEC==1000||KELEC==1200||KELEC==1600||KELEC==1800||KELEC==2000||KELEC==2200||KELEC==2500||KELEC==2800||KELEC==3000||KELEC==3400||KELEC==3600||KELEC==3800||KELEC==4000||KELEC==4200||KELEC==4400||KELEC==4600||KELEC==4800||KELEC==5000||KELEC==5200||KELEC==5400||KELEC==6000||KELEC==6500||KELEC==7000||KELEC==7500||KELEC==8000||KELEC==8500||KELEC==9000||KELEC==9500||KELEC==10500||KELEC==11100){
      		  // ani++;    
           ansetup2++;   
      		 printf("%d %lf\n",ansetup2,t*1.0e9);
           panim();
           gettimeofday(&t_panimend, NULL); 
           tkl=t_panimend.tv_sec; //!new
           info = localtime(&tkl);//!new
           
            printf("%s",asctime (info));//!new
            
           canitimxy=fopen("canitimxy.txt","a"); //! writes the runtime and simulation time correspond to each KELEC, to detect dynamic mesh: dynamic runtime 
           fprintf(canitimxy,"%d ",KELEC);
           fprintf(canitimxy,"%lf ",t*1.0e9);     //! simulation time
           t_panimcal = ((t_panimend.tv_sec - total_start.tv_sec) + ((t_panimend.tv_usec - total_start.tv_usec)/1000000.0));  
           fprintf(canitimxy,"%lf \n",t_panimcal);  //!runtime
           fclose(canitimxy); 
            
           // anim();
           // animE();
	          //bothdenEx();
      	   // bothdenEy();
            
             //  canim();
		      //canimE();  
              }
            //ani++;
           // printf("%d KELEC=%d t=%lf time=%lf\n",ani,KELEC,t,t*1.0e12);
          /*
            denx();
	          diffux();
	          ionfreqx();	
             Ermsx();   
           */     
           /*
            deny();
            diffux();
            diffuy();
            ionfreqx();
            ionfreqy();
            Ermsx();
            Ermsy();
          */
            
          //!%%%%%%% print the temporal profile of Ey-field (Total)  %%%%%%%%%%
          //Eytempo();
         //!%%%%%%% print the temporal profile of density %%%%%%%%%%
         // denytempo();
         //!%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
         // denytempocent();
          //!%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
         // Eytempocent();
          //!%%%%%%%%%%%%%%%%%%%%%
                gettimeofday(&end, NULL);
                t_anim += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
	  
	     //!=============================================================XXXXX
           
            if(KELEC==1)
                fptr=fopen("RES.out","w");
            if(KELEC>1)
            { 
                fptr=fopen("RES.out","a");
                if(fptr!=NULL)
                    fprintf(fptr,"%d %f %f %f %f",n,TIMD,DTAC,ACCEL,dnma);
                else
                    goto l100;
                fclose(fptr);
            }
        }

        l100:

        //!=============================================================XXXXX

        nmod=2*nperdt;

        t=t-dt; //! going back in t=0, for completing the factor time child updates
         for(lup=1;lup<factor;lup++)
         {
            c_n++;
		
            //child_HFIELD();
            
            hipMalloc((void**)&dev_mag, sizeof(child_mag));
            hipMalloc((void**)&dev_c_exs, sizeof(c_exs));
            hipMalloc((void**)&dev_c_eys, sizeof(c_eys));
            hipMalloc((void**)&dev_dtmds, sizeof(dtmds));
            hipMemcpy(dev_mag, child_mag, sizeof(child_mag), hipMemcpyHostToDevice);
            hipMemcpy(dev_exs, c_exs, sizeof(c_exs), hipMemcpyHostToDevice);
            hipMemcpy(dev_eys, c_eys, sizeof(c_eys), hipMemcpyHostToDevice);
            hipMemcpy(dev_dtmds, &dtmds, sizeof(dtmds), hipMemcpyHostToDevice);
            HFIELD<<<(ceil(child_mag->m/32),ceil(child_mag->n/32)),(32,32)>>>(dev_mag,dev_exs,dev_eys,dev_dtmds);
            hipMemcpy(child_mag, dev_mag, sizeof(child_mag), hipMemcpyDeviceToHost);
            hipMemcpy(c_hzi, dev_child_mag->mesh, sizeof(child_mag), hipMemcpyDeviceToHost);

            hipFree(dev_mag);
            hipFree(dev_c_exs);
            hipFree(dev_c_eys);
            hipFree(dev_dtmds);
        child_EFIELD(); 
        // hipMalloc((void**)&dev_elec, sizeof(child_elec));
        // hipMalloc((void**)&dev_x0, sizeof(double));
        // hipMalloc((void**)&dev_OMEG, sizeof(double));
        // hipMalloc((void**)&dev_newt, sizeof(double));
        // hipMalloc((void**)&dev_inv_c, sizeof(double));
        // hipMalloc((void**)&dev_c_dt, sizeof(double));
        // hipMalloc((void**)&dev_sine, sizeof(double));
        // hipMalloc((void**)&dev_sine1, sizeof(double));
        // hipMalloc((void**)&dev_x, sizeof(double));
        // hipMalloc((void**)&dev_c, sizeof(double));
        // hipMalloc((void**)&dev_c_ds, sizeof(double));

        // hipMemcpy(dev_elec, child_elec, sizeof(child_elec), hipMemcpyHostToDevice);
        // hipMemcpy(dev_x0, x0, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_OMEG, OMEG, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_newt, newt, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_inv_c, inv_c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c_dt, c_dt, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c, c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c_ds, c_ds, sizeof(double), hipMemcpyHostToDevice);
        
        
        // child_EFIELD<<<dimGrid, dimBlock>>>(dev_elec,dev_x0,dev_OMEG,dev_newt,dev_inv_c,dev_c_dt,dev_sine,dev_sine1,dev_x,dev_c,dev_c_ds);
        
        // hipMemcpy(sine, dev_sine, sizeof(dev_sine), hipMemcpyDeviceToHost);
        // hipMemcpy(sine1, dev_sine1, sizeof(dev_sine1), hipMemcpyDeviceToHost);

        // hipFree(dev_elec);
        // hipFree(dev_x0);
        // hipFree(dev_OMEG);
        // hipFree(dev_newt);
        // hipFree(dev_inv_c);
        // hipFree(dev_c_dt);
        // hipFree(dev_sine);
        // hipFree(dev_sine1);
        // hipFree(dev_x);
        // hipFree(dev_c);
        // hipFree(dev_c_ds);

            //! interpolate corners, along x along y for E-,H-,electron density for child mesh from parent
            //! inter_value decides the fraction of data contributed from before and updated parent data (provided factor-1 times child update)
            interpolatecorners(inter_value);      
            interpolatex(inter_value);
            interpolatey(inter_value); 
                        
            newt = t + c_dt;            
            KRMS=1;

            //child_RMS(KRMS);

        hipMalloc((void**)&dev_child_elec, sizeof(child_elec));
        hipMalloc((void**)&dev_z1, sizeof(double));
        hipMalloc((void**)&dev_z2, sizeof(double));
        hipMalloc((void**)&dev_inv_nperdt, sizeof(c_inv_nperdt));
        hipMalloc((void**)&dev_ext, sizeof(c_ext));
        hipMalloc((void**)&dev_eyt, sizeof(c_eyt));
        hipMalloc((void**)&dev_erms2, sizeof(c_erms2));
        hipMalloc((void**)&dev_KRMS, sizeof(KRMS));

        hipMemcpy(dev_child_elec, child_elec, sizeof(child_elec), hipMemcpyHostToDevice);
        hipMemcpy(dev_inv_nperdt, &c_inv_nperdt, sizeof(c_inv_nperdt), hipMemcpyHostToDevice);
        hipMemcpy(dev_ext, c_ext, sizeof(c_ext), hipMemcpyHostToDevice);
        hipMemcpy(dev_eyt, c_eyt, sizeof(c_eyt), hipMemcpyHostToDevice);
        //hipMemcpy(dev_ERMSp, c_ERMSp, sizeof(c_ERMSp), hipMemcpyHostToDevice);
        hipMemcpy(dev_erms2, c_erms2, sizeof(c_erms2), hipMemcpyHostToDevice);
        hipMemcpy(dev_KRMS, &KRMS, sizeof(KRMS), hipMemcpyHostToDevice);

        child_RMS<<<(ceil(child_elec->m/32),ceil(child_elec->n/32)),(32,32)>>>(dev_child_elec, dev_z1,dev_z2,dev_inv_nperdt,dev_ext,dev_eyt,dev_erms2,KRMS);
        
        hipMemcpy(c_erms2, dev_erms2, sizeof(dev_erms2), hipMemcpyDeviceToHost);
        if(KRMS==2)
        {
            hipMemcpy(child_elec->mesh, dev_child_elec->mesh, sizeof(dev_child_elec->mesh), hipMemcpyDeviceToHost);
        }
       
        hipFree(dev_child_elec);
        hipFree(dev_z1);
        hipFree(dev_z2);
        hipFree(dev_inv_nperdt);
        hipFree(dev_ext);
        hipFree(dev_eyt);
        hipFree(dev_erms2);
        hipFree(dev_KRMS);

        }

	//!-----------------------(final)
	
  c_n++;

        //child_HFIELD();
        hipMalloc((void**)&dev_mag, sizeof(child_mag));
        hipMalloc((void**)&dev_c_exs, sizeof(c_exs));
        hipMalloc((void**)&dev_c_eys, sizeof(c_eys));
        hipMalloc((void**)&dev_dtmds, sizeof(dtmds));
        hipMemcpy(dev_mag, child_mag, sizeof(child_mag), hipMemcpyHostToDevice);
        hipMemcpy(dev_c_exs, c_exs, sizeof(c_exs), hipMemcpyHostToDevice);
        hipMemcpy(dev_c_eys, c_eys, sizeof(c_eys), hipMemcpyHostToDevice);
        hipMemcpy(dev_dtmds, &dtmds, sizeof(dtmds), hipMemcpyHostToDevice);
        HFIELD<<<(ceil(child_mag->m/32),ceil(child_mag->n/32)),(32,32)>>>(dev_mag,dev_c_exs,dev_c_eys,dev_dtmds);
        hipMemcpy(child_mag, dev_child_mag, sizeof(child_mag), hipMemcpyDeviceToHost);
        hipMemcpy(c_hzi, dev_child_mag->mesh, sizeof(child_mag), hipMemcpyDeviceToHost);

        hipFree(dev_mag);
        hipFree(dev_c_exs);
        hipFree(dev_c_eys);
        hipFree(dev_dtmds);
        child_EFIELD();
        // hipMalloc((void**)&dev_elec, sizeof(child_elec));
        // hipMalloc((void**)&dev_x0, sizeof(double));
        // hipMalloc((void**)&dev_OMEG, sizeof(double));
        // hipMalloc((void**)&dev_newt, sizeof(double));
        // hipMalloc((void**)&dev_inv_c, sizeof(double));
        // hipMalloc((void**)&dev_c_dt, sizeof(double));
        // hipMalloc((void**)&dev_sine, sizeof(double));
        // hipMalloc((void**)&dev_sine1, sizeof(double));
        // hipMalloc((void**)&dev_x, sizeof(double));
        // hipMalloc((void**)&dev_c, sizeof(double));
        // hipMalloc((void**)&dev_c_ds, sizeof(double));

        // hipMemcpy(dev_elec, child_elec, sizeof(child_elec), hipMemcpyHostToDevice);
        // hipMemcpy(dev_x0, x0, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_OMEG, OMEG, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_newt, newt, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_inv_c, inv_c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c_dt, c_dt, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c, c, sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(dev_c_ds, c_ds, sizeof(double), hipMemcpyHostToDevice);
        
        
        // child_EFIELD<<<dimGrid, dimBlock>>>(dev_elec,dev_x0,dev_OMEG,dev_newt,dev_inv_c,dev_c_dt,dev_sine,dev_sine1,dev_x,dev_c,dev_c_ds);
        
        // hipMemcpy(sine, dev_sine, sizeof(dev_sine), hipMemcpyDeviceToHost);
        // hipMemcpy(sine1, dev_sine1, sizeof(dev_sine1), hipMemcpyDeviceToHost);  

        // hipFree(dev_elec);
        // hipFree(dev_x0);
        // hipFree(dev_OMEG);
        // hipFree(dev_newt);
        // hipFree(dev_inv_c);
        // hipFree(dev_c_dt);
        // hipFree(dev_sine);
        // hipFree(dev_sine1);
        // hipFree(dev_x);
        // hipFree(dev_c);
        // hipFree(dev_c_ds);
        //! interpolate corners, along x along y for E-,H-,electron density for child mesh from parent
        //! inter_value decides the fraction of data contributed only from updated parent data (provided on the factor th update of child)   
        interpolatecorners(1.0);
        interpolatex(1.0);
        interpolatey(1.0);
       	
       
        t=t+dt;
        newt=t;
        
        KRMS=1;
        if(fmod(c_n,c_nperdt)==0) {
                   
            KRMS=2;
        }
        //child_RMS(KRMS);

        hipMalloc((void**)&dev_child_elec, sizeof(child_elec));
        hipMalloc((void**)&dev_z1, sizeof(double));
        hipMalloc((void**)&dev_z2, sizeof(double));
        hipMalloc((void**)&dev_inv_nperdt, sizeof(c_inv_nperdt));
        hipMalloc((void**)&dev_ext, sizeof(c_ext));
        hipMalloc((void**)&dev_eyt, sizeof(c_eyt));
        hipMalloc((void**)&dev_erms2, sizeof(c_erms2));
        hipMalloc((void**)&dev_KRMS, sizeof(KRMS));

        hipMemcpy(dev_child_elec, child_elec, sizeof(child_elec), hipMemcpyHostToDevice);
        hipMemcpy(dev_inv_nperdt, &c_inv_nperdt, sizeof(c_inv_nperdt), hipMemcpyHostToDevice);
        hipMemcpy(dev_ext, c_ext, sizeof(c_ext), hipMemcpyHostToDevice);
        hipMemcpy(dev_eyt, c_eyt, sizeof(c_eyt), hipMemcpyHostToDevice);
        //hipMemcpy(dev_ERMSp, c_ERMSp, sizeof(c_ERMSp), hipMemcpyHostToDevice);
        hipMemcpy(dev_erms2, c_erms2, sizeof(c_erms2), hipMemcpyHostToDevice);
        hipMemcpy(dev_KRMS, &KRMS, sizeof(KRMS), hipMemcpyHostToDevice);

        child_RMS<<<(ceil(child_elec->m/32),ceil(child_elec->n/32)),(32,32)>>>(dev_child_elec, dev_z1,dev_z2,dev_inv_nperdt,dev_ext,dev_eyt,dev_erms2,dev_KRMS);
        
        hipMemcpy(c_erms2, dev_erms2, sizeof(dev_erms2), hipMemcpyDeviceToHost);
        if(KRMS==2)
        {
            hipMemcpy(child_elec->mesh, dev_child_elec->mesh, sizeof(dev_child_elec->mesh), hipMemcpyDeviceToHost);
        }
       
        hipFree(dev_child_elec);
        hipFree(dev_z1);
        hipFree(dev_z2);
        hipFree(dev_inv_nperdt);
        hipFree(dev_ext);
        hipFree(dev_eyt);
        hipFree(dev_erms2);
        hipFree(dev_KRMS);
        

        if(fmod(c_n,c_nperdt*nmaxwell)==0){
            if(icpling!=0){
             child_ELEC_DENS();     //!  each child period update the child electron density

            }
            else{
                DTAC=1.0/FREQ;
                TIMD=(double)(c_n)*inv_nperdt*nmaxwell*DTAC/factor;
            }
            c_KELEC++;
        }

	        c2p();         //! only after the child data updated between a parent update copy the fine (child) data back to parent
                          //! to avoid mismatch in accuracy . copy cpb(child-parent) boundary 
                 
                    
    }while(1);

        //! prints the calculation time for the each of the subroutines (remove the rank as it is for MPI code)
        fptr2 = fopen("mes_result.csv","a");
        fprintf(fptr2,"Processor.Rank ");            
        fprintf(fptr2,"Cal.EFIELD ");            
        fprintf(fptr2,"Cal.Child_EFIELD ");            
        fprintf(fptr2,"Cal.HFIELD ");           
        fprintf(fptr2,"Cal.Child_HFIELD ");        
        fprintf(fptr2,"Cal.ELEC_DENS " );        
        fprintf(fptr2,"Cal.CHILD_ELEC_DENS ");        
        fprintf(fptr2,"Cal.RMS ");           
        fprintf(fptr2,"Cal.CHILD_RMS ");
        fprintf(fptr2,"Cal.c2p ");
        fprintf(fptr2,"Cal.interpolatex ");
        fprintf(fptr2,"Cal.interpolatexinitial ");
        fprintf(fptr2,"Cal.interpolatexnew ");
        fprintf(fptr2,"Cal.interpolateynew ");
        fprintf(fptr2,"Cal.anim ");
        fprintf(fptr2,"Total.time ");
        fprintf(fptr2,"Program.time\n");                
        fclose(fptr2);
        
       fptr2 = fopen("mes_result.csv","a");
        fprintf(fptr2,"%d ",rank);            
        fprintf(fptr2,"%f ",t_cal_efield);            
        fprintf(fptr2,"%f ",t_cal_child_efield);            
        fprintf(fptr2,"%f ",t_cal_hfield);           
        fprintf(fptr2,"%f ",t_cal_child_hfield);           
        fprintf(fptr2,"%f ",t_cal_elec_dens);            
        fprintf(fptr2,"%f ",t_cal_child_elec_dens);            
        fprintf(fptr2,"%f ",t_cal_rms);           
        fprintf(fptr2,"%f ",t_cal_child_rms);           
        fprintf(fptr2,"%f ",t_cal_c2p);           
        fprintf(fptr2,"%f ",t_cal_interpolatex); 
        fprintf(fptr2,"%f ",t_cal_interpolatexinitial);
        fprintf(fptr2,"%f ",t_cal_interpolatexnew);
        fprintf(fptr2,"%f ",t_cal_interpolateynew);
        fprintf(fptr2,"%f ",t_cal_anim);            
        fprintf(fptr2,"%f ",t1);                       
        gettimeofday(&program_end,NULL);    
        t1 = ((program_end.tv_sec - program_start.tv_sec) + ((program_end.tv_usec - program_start.tv_usec)/1000000.0));
        fprintf(fptr2,"%f\n",t1);                       
        fclose(fptr2);      

      printf("%e %d %d\n", dt,nx,ny); 
      printf("OUTPUT 2\n");
      gettimeofday(&total_end, NULL);
      printf("EFIELD time: %f s\n", t_efield_hfield);
      printf("ELEC_DENS time: %f s\n", t_elec_dens);
      printf("RMS time: %f s\n", t_rms);
      printf("Zero time: %f s\n", t_zero);
      printf("Anim time: %f s\n", t_anim);
      printf("Total time: %f s\n", ((total_end.tv_sec - total_start.tv_sec) + ((total_end.tv_usec - total_start.tv_usec)/1000000.0)));
      
  	free_all();
    return 0;
}

void EFIELD()
{    
     gettimeofday(&begin,NULL);
    for(i=0; i<root_elec->m;i++)
    {
    	x=i*ds;          
        sine=0.0;
        sine1=0.0;
        if(x<=(x0+c*t))
        { 
            sine = sin(OMEG*(t-(x-x0)*inv_c));  //!forward wave
        }
        if(x<=(x0+c*(t+dt))) 
        {   
            sine1 = sin(OMEG*(t+dt-(x-x0)*inv_c));  //!forward wave
        }

        

        for(j=0;j<root_elec->n;j++)
        {
            
            eyi[i][j] =   E0*(sine);  //! incident field () at t
            eyi1[i][j] =  E0*(sine1); //! incident field () at t+dt
         
        }

        for(j=0;j<root_elec->n;j++)
        {
            omp2x=(root_den->mesh[i][j]+root_den->mesh[i+1][j])*const3;
            betax=omp2x*const4;
            const5x=1.0/(1.0+betax);
            const6x=1.0-betax;
            if(j>0)
            {
            	extk=ext[i][j];
                exs_old[i][j] = exs[i][j];
             //! Scattered E-field x update depends on previous E-data, previous H-data (top/bottom), density data (space:left/ right),velocity (same location)     
                exs[i][j]=const5x*( exs[i][j]*(const6x)+qe*(root_den->mesh[i][j]+root_den->mesh[i+1][j])*vx[i][j]*const7
                                     -(exi[i][j]+exi1[i][j])*betax+(root_mag->mesh[i][j]-root_mag->mesh[i][j-1])*dteds); 

                vx_old[i][j] = vx[i][j];
				ext[i][j]=exs[i][j]+exi1[i][j];     //! Total field = inci + scattered 
                vx[i][j]=vx[i][j]*alpha - qmdt*(ext[i][j]+extk)*const8;
        
               
            }
            if(i>0)
            {

                omp2y=(root_den->mesh[i][j]+root_den->mesh[i][j+1])*const3;
                betay=omp2y*const4;
                const5y=1.0/(1.0+betay);
                const6y=1.0-betay;

                eytk=eyt[i][j];
                eys_old[i][j] = eys[i][j];      //! reqd for interpolation
                //!Scattered E-field y update depends on previous E-data, previous H-data (left/right), density data (space:top/ bottom),velocity (same location)
                eys[i][j]=const5y*(eys[i][j]*(const6y)+qe*(root_den->mesh[i][j]+root_den->mesh[i][j+1])*vy[i][j]*const7
                                 -(eyi[i][j]+eyi1[i][j])*betay-(root_mag->mesh[i][j]-root_mag->mesh[i-1][j])*dteds);


                eyt[i][j]=eys[i][j]+eyi1[i][j];    //! Total field = inci + scattered 
                vy_old[i][j] = vy[i][j];        //! reqd for interpolation
                vy[i][j]=vy[i][j]*alpha - qmdt*(eyt[i][j]+eytk)*const8;     //! velocity update
            }
        }
        MR_MUR(i);
    }
    gettimeofday(&end,NULL);    
    t_cal_efield += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

  //int dev_child_elec,dev_x0,dev_OMEG,dev_newt,dev_inv_c,dev_c_dt,dev_sine,dev_sine1,dev_x 

// __global__ EFIELD(struct node * elec,double x0,double OMEG,double newt,double inv_c,double c_dt,double sine,double sine1,double x,double c)
// {
//     int i = blockIdx.x*blockDim.x+threadIdx.x;
//     if(i<elec->m)
//     {
//         x=i*ds;          
//          sine=0.0;
//          sine1=0.0;
//          if(x<=(x0+c*t))
//          { 
//              sine = sin(OMEG*(t-(x-x0)*inv_c));  //!forward wave
//          }
//          if(x<=(x0+c*(t+dt))) 
//          {   
//              sine1 = sin(OMEG*(t+dt-(x-x0)*inv_c));  //!forward wave
//          }
//     }

// }

// __global__ child_EFIELD(struct node * child_elec,double x0,double OMEG,double newt,double inv_c,double c_dt,double sine,double sine1,double x,double c,double c_ds)
// {
//     int i = blockIdx.x*blockDim.x+threadIdx.x;
//     if(i<elec->m)
//     {
//         x=(i+factor*child_elec->locx)*c_ds;          
//         sine=0.0;
//         sine1=0.0;
// 	        if(x<=(x0+c*(newt)))
// 	        { 
// 	            sine = sin(OMEG*(newt-(x-x0)*inv_c));    //!forward wave
	           
// 	        }
// 	        if(x<=(x0+c*(newt+c_dt))) 
// 	        {   
	        	
// 	            sine1 = sin(OMEG*(newt+c_dt-(x-x0)*inv_c));    //!forward wave
	            
// 	        }
	    
//     }
// }
void child_EFIELD()
{
	
   gettimeofday(&begin,NULL);
	for(i=0; i<child_elec->m;i++)
    {
    	x=(i+factor*child_elec->locx)*c_ds;          
        sine=0.0;
        sine1=0.0;
	        if(x<=(x0+c*(newt)))
	        { 
	            sine = sin(OMEG*(newt-(x-x0)*inv_c));    //!forward wave
	           
	        }
	        if(x<=(x0+c*(newt+c_dt))) 
	        {   
	        	
	            sine1 = sin(OMEG*(newt+c_dt-(x-x0)*inv_c));    //!forward wave
	            
	        }
	    
    	    
          for(j=0;j<child_elec->n;j++)
    	    {
      	    
              c_eyi[i][j] =   E0*(sine);    //! incident field ( forward direction) at newt
    	        
    	        c_eyi1[i][j] =  E0*(sine1);    //! incident field ( forward direction) at newt+c_dt
    	    }

        for(j=0;j<child_elec->n;j++)
        {
            omp2x=(child_den->mesh[i][j]+child_den->mesh[i+1][j])*const3;
            betax=omp2x*c_const4;
            const5x=1.0/(1.0+betax);
            const6x=1.0-betax;
            if(j>0)
            {
                extk=c_ext[i][j];
                c_exs[i][j]=const5x*( c_exs[i][j]*(const6x)+qe*(child_den->mesh[i][j]+child_den->mesh[i+1][j])*c_vx[i][j]*c_const7 
                                     -(c_exi[i][j]+c_exi1[i][j])*betax+(child_mag->mesh[i][j]-child_mag->mesh[i][j-1])*c_dteds); //const7 is divided by 2 because of the dt factor.

                c_exold[i][j]=c_exs[i][j];    //! reqd when expanding the child mesh to copy the previous mesh data
                
                c_ext[i][j]=c_exs[i][j]+c_exi1[i][j];   //! Total field (child)= inci + scattered 


                c_vx[i][j]=c_vx[i][j]*c_alpha - c_qmdt*(c_ext[i][j]+extk)*c_const8;
               
                c_vxold[i][j]=c_vx[i][j];     //! reqd when expanding the child mesh to copy the previous mesh data
            }
            if(i>0)
            {

                omp2y=(child_den->mesh[i][j]+child_den->mesh[i][j+1])*const3;
                betay=omp2y*c_const4;
                const5y=1.0/(1.0+betay);
                const6y=1.0-betay;

                eytk=c_eyt[i][j];

                double v = c_eys[i][j]*(const6y);

                c_eys[i][j]=const5y*(c_eys[i][j]*(const6y)+qe*(child_den->mesh[i][j]+child_den->mesh[i][j+1])*c_vy[i][j]*c_const7
                                 -(c_eyi[i][j]+c_eyi1[i][j])*betay-(child_mag->mesh[i][j]-child_mag->mesh[i-1][j])*c_dteds);
               	            
                c_eyold[i][j]=c_eys[i][j];  //! reqd when expanding the child mesh to copy the previous mesh data

                c_eyt[i][j]=c_eys[i][j]+c_eyi1[i][j];  //! Total field (child)= inci + scattered 

                c_vy[i][j]=c_vy[i][j]*c_alpha - c_qmdt*(c_eyt[i][j]+eytk)*c_const8;
                
                c_vyold[i][j]=c_vy[i][j];   //! reqd when expanding the child mesh to copy the previous mesh data
            }
        }
    }
    gettimeofday(&end,NULL);    
    t_cal_child_efield += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

// void HFIELD()
// {
//     gettimeofday(&begin,NULL); 
//     for(i=0; i<root_mag->m; i++)
//     {
//     	for(j=0; j<root_mag->n; j++)
//     	{
//     		hzi[i][j] = root_mag->mesh[i][j]; //! if we give H-excitation rather E-excitation 
//         //! H-field (parent) depends on E-field x:  top/bottom and E-field y: left/right previous (before E-field updates then H-field)
//    		root_mag->mesh[i][j]+= (-(eys[i+1][j]-eys[i][j])+(exs[i][j+1]-exs[i][j]))*dtmds;
//     	}
//     }
//     gettimeofday(&end,NULL);    
//     t_cal_hfield += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
// }   


// void child_HFIELD()
// {
//    gettimeofday(&begin,NULL);
// 	for(i=0; i<child_mag->m; i++)
//     {
//     	for(j=0; j<child_mag->n; j++)
//     	{
//     		c_hzi[i][j] = child_mag->mesh[i][j];
//          //! H-field (parent) depends on E-field x:  top/bottom and E-field y: left/right previous (before E-field updates then H-field)
//     		child_mag->mesh[i][j]+= (-(c_eys[i+1][j]-c_eys[i][j])+(c_exs[i][j+1]-c_exs[i][j]))*c_dtmds;
        
//         c_hzold[i][j]=child_mag->mesh[i][j];  
//     	}
//     }
//     gettimeofday(&end,NULL);    
//     t_cal_child_hfield += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
// }


void RMS(int k)
{
    //printf("In RMS\n");
    gettimeofday(&begin,NULL);
    double z1,z2;
    
    int i;
    for(i=1;i<root_elec->m;i++)
    {
        for(j=0;j<root_elec->n;j++)
        {
            z1=(ext[i][j]*ext[i][j]+ext[i-1][j]*ext[i-1][j])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
            z2=(eyt[i][j]*eyt[i][j]+eyt[i][j-1]*eyt[i][j-1])*.5f;   //! avg of the two scattered field is required (E_eff) for the density update
            ERMSp[i][j] = erms2[i][j];
            erms2[i][j]=erms2[i][j]+(z1+z2)*inv_nperdt;     //! time updates and averages (parent)
        }
    }

    if(k==2)
    {
        int j;
        for(i=0;i<root_elec->m;i++)
        {
            for(j=0;j<root_elec->n;j++)
            {
            	if (erms2[i][j]<0)
            	{
            		printf("Alert!!\n");
            	}
                root_elec->mesh[i][j] = sqrt(erms2[i][j]);  //! completes a period and then squre root the time avg data (parent)
                erms2[i][j]=0.0f;
            } 
        }
    }
    gettimeofday(&end,NULL);
    t_cal_rms += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void child_RMS(int k)
{
	// printf("In RMS\n");
   gettimeofday(&begin,NULL); 
	double z1,z2;

    int i;
    for(i=1;i<child_elec->m;i++)
    {
        for(j=1;j<child_elec->n;j++)
        {
            z1=(c_ext[i][j]*c_ext[i][j]+c_ext[i-1][j]*c_ext[i-1][j])*.5f; //! avg of the two scattered field is required (E_eff) for the density update
            z2=(c_eyt[i][j]*c_eyt[i][j]+c_eyt[i][j-1]*c_eyt[i][j-1])*.5f; //! avg of the two scattered field is required (E_eff) for the density update
            c_erms2[i][j]=c_erms2[i][j]+(z1+z2)*c_inv_nperdt;  //! time updates and averages (child)
            
        }
    }

    if(k==2)
    {
        int j;
        
        for(i=0;i<child_elec->m;i++)
        {
            for(j=0;j<child_elec->n;j++)
            {
            	if (c_erms2[i][j]<0)
            	{
            		printf("Alert!!\n");
            	}
                child_elec->mesh[i][j] = sqrt(c_erms2[i][j]);  //! completes a period and then squre root the time avg data (child)
                c_erms2[i][j]=0.0f;
            } 
        }
    }
    gettimeofday(&end,NULL);    
    t_cal_child_rms += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void MR_MUR(int row)
{
    //! idea for the MUR boundary is to provide the E-field value at the four boundaries (top (ny)/bottom(0) and left (0)/right(nx))
    //! the values present at the +1 from left, -1 from right and -1from top and +1 from bottom from two time updates
   //! To avoid discontinuity due to truncation of the infinite boundary and absorb the scattered waves
    double csym;
    csym=1.0;
    if(IABSOR==2)
        csym=0.0;

    int column=0;
    if(row==1)
    {   
        for(column=0;column<ny;column++){
            eys[0][column]=eys1[1][column]+c1*(eys[1][column]-eys1[0][column]);
            eys1[0][column]=eys[0][column];  
            eys1[1][column]=eys[1][column];      
        } 
    }
    if(row==nx-1)
    {
        for(column=0;column<ny;column++)
        {
	        eys[nx][column]=eys1[nx-1][column]+c1*(eys[nx-1][column]-eys1[nx][column]);
	        eys1[nx][column]=eys[nx][column];
	        eys1[nx-1][column]=eys[nx-1][column];
        }   
    }
    exs[row][0]=exs1[row][1] +csym*c1*(exs[row][1]-exs1[row][0]);
    exs1[row][0]=exs[row][0];
    exs1[row][1]=exs[row][1];
    exs[row][ny]=exs1[row][ny-1] +csym*c1*(exs[row][ny-1]-exs1[row][ny]);
    exs1[row][ny]=exs[row][ny];
    exs1[row][ny-1]=exs[row][ny-1];
}

void child_MR_MUR_0(int row)
{
    double csym;
    csym=1.0;
    if(IABSOR==2)
        csym=0.0;

    int column=0;
    if(row==1)
    {   
        column=0;
        c_eys[0][column]=(c_eys1[1][column]+c_c1*(c_eys[1][column]-c_eys1[0][column]));
        c_eys1[0][column]=c_eys[0][column];  
        c_eys1[1][column]=c_eys[1][column];      
    }
    if(row==child_elec->m-1)
    {
        column=0;
        c_eys[child_elec->m][column]=(c_eys1[child_elec->m-1][column]+c_c1*(c_eys[child_elec->m-1][column]-c_eys1[child_elec->m][column]));
        c_eys1[child_elec->m][column]=c_eys[child_elec->m][column];
        c_eys1[child_elec->m-1][column]=c_eys[child_elec->m-1][column];
    }
    c_exs[row][0]=(c_exs1[row][1] +csym*c_c1*(c_exs[row][1]-c_exs1[row][0]));
    c_exs1[row][0]=c_exs[row][0];
    c_exs1[row][1]=c_exs[row][1];
}

void child_MR_MUR_1(int row)
{
    double csym;
    csym=1.0;
    if(IABSOR==2)
        csym=0.0;

    int column=0;
    if(row==child_elec->m-1)
    {
        for(column=0;column<child_elec->n;column++)
        {
	        c_eys[child_elec->m][column]=(c_eys1[child_elec->m-1][column]+c_c1*(c_eys[child_elec->m-1][column]-c_eys1[child_elec->m][column]));
	        c_eys1[child_elec->m][column]=c_eys[child_elec->m][column];
	        c_eys1[child_elec->m-1][column]=c_eys[child_elec->m-1][column];
        }   
	    c_exs[row][0]=(c_exs1[row][1] +csym*c_c1*(c_exs[row][1]-c_exs1[row][0]));
	    c_exs1[row][0]=c_exs[row][0];
	    c_exs1[row][1]=c_exs[row][1];
	    c_exs[row][child_elec->n]=(c_exs1[row][child_elec->n-1] +csym*c_c1*(c_exs[row][child_elec->n-1]-c_exs1[row][child_elec->n]));
	    c_exs1[row][child_elec->n]=c_exs[row][child_elec->n];
	    c_exs1[row][child_elec->n-1]=c_exs[row][child_elec->n-1];
	}
}

void child_MR_MUR_2(int row)
{
    double csym;
    csym=1.0;
    if(IABSOR==2)
        csym=0.0;

    int column=0;
    if(row==1)
    {   
        column=child_elec->n-1;
        c_eys[0][column]=(c_eys1[1][column]+c_c1*(c_eys[1][column]-c_eys1[0][column]));
        c_eys1[0][column]=c_eys[0][column];  
        c_eys1[1][column]=c_eys[1][column];      
    }
    if(row==child_elec->m-1)
    {
        column=child_elec->n-1;
        c_eys[child_elec->m][column]=(c_eys1[child_elec->m-1][column]+c_c1*(c_eys[child_elec->m-1][column]-c_eys1[child_elec->m][column]));
        c_eys1[child_elec->m][column]=c_eys[child_elec->m][column];
        c_eys1[child_elec->m-1][column]=c_eys[child_elec->m-1][column];
    }
    c_exs[row][child_elec->n]=(c_exs1[row][child_elec->n-1] +csym*c_c1*(c_exs[row][child_elec->n-1]-c_exs1[row][child_elec->n]));
    c_exs1[row][child_elec->n]=c_exs[row][child_elec->n];
    c_exs1[row][child_elec->n-1]=c_exs[row][child_elec->n-1];
}

void child_MR_MUR_3(int row)
{
    double csym;
    csym=1.0;
    if(IABSOR==2)
        csym=0.0;

    int column=0;
    if(row==1)
    {   
        for(column=0;column<child_elec->n;column++){
            c_eys[0][column]=(c_eys1[1][column]+c_c1*(c_eys[1][column]-c_eys1[0][column]));
            c_eys1[0][column]=c_eys[0][column];  
            c_eys1[1][column]=c_eys[1][column];      
        } 
    }
    
    if(row ==1)
    {
	    c_exs[row][0]=(c_exs1[row][1] +csym*c_c1*(c_exs[row][1]-c_exs1[row][0]));
	    c_exs1[row][0]=c_exs[row][0];
	    c_exs1[row][1]=c_exs[row][1];
	    c_exs[row][child_elec->n]=(c_exs1[row][child_elec->n-1] +csym*c_c1*(c_exs[row][child_elec->n-1]-c_exs1[row][child_elec->n]));
	    c_exs1[row][child_elec->n]=c_exs[row][child_elec->n];
	    c_exs1[row][child_elec->n-1]=c_exs[row][child_elec->n-1];
    }
}

double FIONIZ(double EE,double PR)
{
    //printf("In FIONIZ\n");
    double fioniz,ARG,VD;
    amu=QSM/FNUM*PRESSURE/PR;
    VD=amu*EE*PR;
    
    if(EE>2.0e4)
    {
	    ARG=BB1/EE;
	    fioniz=AA1*PR*exp(-ARG)*VD; 
    } 
    else if(EE<5.0e3)
    {
	    ARG=BB3*(1.0/EE-1.0/EE3);
	    fioniz=AA3*PR*(exp(-ARG)-1.0)*VD;
    } 
    else
    {
	    ARG=BB2/EE;
	    fioniz=AA2*PR*exp(-ARG)*VD;
    }
    return fioniz;
}

void ELEC_DENS()
{
	// printf("In parent elec dens %d\n",n);
    // printf("In ELEC_DENS\n");
    /*
    routine to calculate electron DENsity
    free diffusion + ambipolar dIFfusion + ioniZation
    */
    gettimeofday(&begin,NULL); 
    
    int ie,ied,je,jed,iii;
    double fioniz,aad,taumij;
    double coref,ee,cf,da,dac,fnui,fnua;
    double omgc2,rec1;
    double d0,dimax,ecm;
    double frqij,dtacmax,tcycle;

    //!=============================================================
    dnma=0.0;   //! max density at grids
    dimax=0.0;

    ACCEL=naccel; 
    if(ACCEL<=1) 
    ACCEL=1.0;

    coref=FNUM/sqrt(nu2omeg2);
    amu=EMOB;

    //c---------------------------------------------------------
    int i;
    
    
    for(i=1;i<nx;i++)
    {
        for(j=1;j<ny;j++)
        { 
            denp[i][j]=root_den->mesh[i][j];
            ee=root_elec->mesh[i][j];
            ee=ee/PRESSURE*coref;
            
            //c------------ calculation of ionization frequency---------
            frqio[i][j]=FIONIZ(ee,PRESSURE);    //ionization frequency
           
            //c---------------------------------------------------------
            
            cf=1.0;
            if(root_den->mesh[i][j]<=0.1) 
                cf=0.0;
            //          FNUIMAX=AMAX1(FNUIMAX,cf*frqio(I,J))
            frqij=frqio[i][j];

            //c----- calculation of diffusion coefficient--------------

            EDIF=dife;
            if(frqij<0)
                frqij=0.0;        

            if(cene==0.0){    //use formula for ETEM calc. Edif = diffusion of electron (free)
                ecm=ee*0.01;    
                ETEM=0.1+2.0/3.0*pow((0.0021*ecm*(91.0+ecm)),0.3333);
                EDIF=EMOB*ETEM;
            }

            if(cene<0.0)    //! to calculate the effective diffusion coefficient Edif, using a fixed ETEM=2 eV
            {
                taumij=eps0/(qe*(root_den->mesh[i][j]+1.0)*amu);
                aad=frqij*taumij;
                EDIF=(aad*dife+difa)/(aad+1.0);
            }
        
            //c---------------------------------------------------------              
            DIFFUSION[i][j]=EDIF;
         
            dimax=Max(dimax,DIFFUSION[i][j]);
            
        }
   	}
    //c---------------------------------------------------------              

    //c------- time step for fluid equation------------
    dtacmax=0.20*ds*ds/dimax;
    tcycle=1.0/FREQ;
    DTAC=Min(dtacmax, ndifmax*tcycle*ACCEL);

    for(i=1;i<root_den->m;i++)
    {
        for(j=1;j<root_den->n;j++)
        {
            da=DIFFUSION[i][j];
            dac=da*DTAC/(ds*ds);
            
            rec1=RECOMB*root_den->mesh[i][j]*DTAC;
            
            //c------------ ionization and attachment frequency-------------
            fnua=0.0;
            fnui=frqio[i][j];
            
            if(fnui<0.0)
            {
                fnua=-fnui;
                fnui=0.0;
            }
           
                if(isnan(denp[i][j]*exp(fnui*DTAC)))
            	{
            		printf("entered here%d\t%d\t%e\t%d\n",i,j,root_elec->mesh[i][j],n);
                	exit(0);
                }

            //c---------- Density equation updates------------------------

            //! parent mesh electron density update requires the previous densities (left, right, top and bottom and the center one)      
            root_den->mesh[i][j]=denp[i][j]*exp(fnui*DTAC)+dac*(denp[i+1][j]+denp[i-1][j]+denp[i][j+1]+denp[i][j-1]-4.0*denp[i][j]);
            root_den->mesh[i][j]=root_den->mesh[i][j]/(1.0+rec1+fnua*DTAC);

            if(root_den->mesh[i][j]<=1e-15)
                root_den->mesh[i][j]=root_den->mesh[i][j]*0.0;

            dnma=Max(dnma,root_den->mesh[i][j]);
        }
    
	}
	
    //c---------------Actual time calculation -------------------------
    TIMD=TIMD+DTAC;
    //c------------------------------------------------------------
    
    omgc2=qe*qe*inv_cmasse/eps0*dnma;
    PARC=omgc2/(pow(OMEG,2)+pow(FNUM,2));
  
    gettimeofday(&end,NULL);    
    t_cal_elec_dens += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void child_ELEC_DENS()
{
	// printf("In child elec dens %d\n",n);
  gettimeofday(&begin,NULL);
	int ie,ied,je,jed,iii;
    double fioniz,aad,taumij;
    double coref,ee,cf,da,dac,fnui,fnua;
    double omgc2,rec1;
    double d0,dimax,ecm;
    double frqij,dtacmax,tcycle;

    //!=============================================================
    dnma=0.0;   //! max density at grids
    dimax=0.0;

    ACCEL=naccel; 
    if(ACCEL<=1) 
    ACCEL=1.0;

    coref=FNUM/sqrt(nu2omeg2);
    amu=EMOB;

    //c---------------------------------------------------------
    int i;
    
    
    for(i=1;i<child_den->m;i++)
    {
        for(j=1;j<child_den->n;j++)
        { 
        	c_denp[i][j]=child_den->mesh[i][j];
            ee=child_elec->mesh[i][j];
            ee=ee/PRESSURE*coref;
            
            //c------------ calculation of ionization frequency---------
            c_frqio[i][j]=FIONIZ(ee,PRESSURE);   //! the ionization frequency
            //c---------------------------------------------------------
            
            cf=1.0;
            if(child_den->mesh[i][j]<=0.1) 
                cf=0.0;
            //          FNUIMAX=AMAX1(FNUIMAX,cf*frqio(I,J))
            frqij=c_frqio[i][j];

            //c----- calculation of diffusion coefficient--------------

            EDIF=dife;
            if(frqij<0)
                frqij=0.0;        

            if(cene==0.0){
                ecm=ee*0.01;    
                ETEM=0.1+2.0/3.0*pow((0.0021*ecm*(91.0+ecm)),1/3);
                EDIF=EMOB*ETEM;
            }

            if(cene<0.0)
            {
                taumij=eps0/(qe*(child_den->mesh[i][j]+1.0)*amu);
                aad=frqij*taumij;
                EDIF=(aad*dife+difa)/(aad+1.0);
            }
        
            //c---------------------------------------------------------              
            c_DIFFUSION[i][j]=EDIF;
         
            dimax=Max(dimax,c_DIFFUSION[i][j]);
            
        }
   	}
    //c---------------------------------------------------------              

    //c------- time step for fluid equation------------
    dtacmax=0.20*c_ds*c_ds/dimax;
    tcycle=1.0/FREQ;
    DTAC=Min(dtacmax, ndifmax*tcycle*ACCEL);

    for(i=1;i<child_den->m;i++)
    {
        for(j=1;j< child_den->n;j++)
        {
            da=c_DIFFUSION[i][j];
            dac=da*DTAC/(c_ds*c_ds);
            
            rec1=RECOMB*child_den->mesh[i][j]*DTAC;
            
            //c------------ ionization and attachment frequency-------------
            fnua=0.0;
            fnui=c_frqio[i][j];
            if(fnui<0.0)
            {
                fnua=-fnui;
                fnui=0.0;
            }

            if(isnan(c_denp[i][j]*exp(fnui*DTAC)))
            {
            	printf("Child, n = %d\n", n);
            	exit(0);
            }

            //c---------- Density equation updates------------------------
            //! child mesh electron density update requires the previous densities (left, right, top and bottom and the center one)      
            child_den->mesh[i][j]=c_denp[i][j]*exp(fnui*DTAC)+dac*(c_denp[i+1][j]+c_denp[i-1][j]+c_denp[i][j+1]+c_denp[i][j-1]-4.0*c_denp[i][j]);
            


            child_den->mesh[i][j]=child_den->mesh[i][j]/(1.0+rec1+fnua*DTAC);
            
            c_denpold[i][j]=child_den->mesh[i][j];      //! reqd when expanding the child mesh to copy the previous mesh data
            
            
            if(child_den->mesh[i][j]<=1e-15)
                child_den->mesh[i][j]=child_den->mesh[i][j]*0.0;

            dnma=Max(dnma,child_den->mesh[i][j]);
        }
    
	}

    //c---------------Actual time calculation -------------------------
    TIMD=TIMD+DTAC;
    //c------------------------------------------------------------
    
    omgc2=qe*qe*inv_cmasse/eps0*dnma;
    PARC=omgc2/(pow(OMEG,2)+pow(FNUM,2));
    
    gettimeofday(&end,NULL);    
    t_cal_child_elec_dens += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void SETUP()
{
    // printf("In SETUP\n");
    static int k;
    static double ardiy,ardix,dinig;
    static double xd0,yd0,xxi,yyj;
    static double c_ardiy,c_ardix,c_dinig;
    static double c_xd0,c_yd0,c_xxi,c_yyj;
    static int xtotal,ytotal;
    //static int xend,yend,xstar,ystar;
//!---------------------------divisible by 4(child data)-----    
    //xend=(9*nx/10);yend=(9*ny/10);xstar=(1*nx/2);ystar=(1*ny/10);
    xend=(xe*nx);yend=(ye*ny);xstar=(xs*nx);ystar=(ys*ny);
    /*  
     xend = 0.65*nx;
    xstar = 0.35*nx;
    yend = 0.9*ny;
    ystar = 0.1*ny;
  */
  /*
     xend = 0.8*nx;
    xstar = 0.2*nx;
    yend = 0.9*ny;
    ystar = 0.1*ny;
  */
   /* 
    xend = 0.95*nx;
    xstar = 0.05*nx;
    yend = 0.9*ny;
    ystar = 0.1*ny;
    */
    
   printf("Before,xend=%d\tyend=%d\txstar=%d\tystar=%d\t\n",xend,yend,xstar,ystar);
	
    xtotal=(xend-xstar)*factor;
    ytotal=(yend-ystar)*factor;
    printf("after,xend=%d\tyend=%d\txstar=%d\tystar=%d\t\n",xend,yend,xstar,ystar);
//!--------------------------------------------------------------
	
		
  root_elec = newnode(nx, ny, 0, 0, 0);
	child_elec = newnode(xtotal, ytotal, xstar, ystar, 1);
	
  root_elec->mesh = root_mesh_elec;
	child_elec->mesh = child_mesh_elec;

	root_elec->children[0] = child_elec;
	root_elec->parent = NULL;

	child_elec->parent = root_elec;


	root_mag = newnode(nx, ny, 0, 0, 0);
	child_mag = newnode(xtotal, ytotal, xstar, ystar, 1);

	root_mag->mesh = root_mesh_mag;
	child_mag->mesh = child_mesh_mag;

	root_mag->children[0] = child_mag;
	root_mag->parent = NULL;

	child_mag->parent = root_mag;


	root_den = newnode(nx, ny, 0, 0, 0);
	child_den = newnode(xtotal, ytotal, xstar, ystar, 1);

	root_den->mesh = root_mesh_den;
	child_den->mesh = child_mesh_den;

	root_den->children[0] = child_den;
	root_den->parent = NULL;

	child_den->parent = root_den;


    dt=1.0/(double)(nperdt)/FREQ;
    ds=c/(double)(nlamb)/FREQ;

    c_dt=dt/factor;
    c_ds=ds/factor;

    OMEG=2.0*pi*FREQ;
    QSM=qe/cmasse;
    e2_epsm=qe*qe/eps0/cmasse;
    
    /*
    !  AIR DAT
    !  FNUM=electron-neutral coll frequency
    !  RECOMB=electron-ion recombination coefficient
    !  EMOB=electron mobility
    !  ETEM=electron temperature
    !  DIFE=electron diffusion coeff
    */

    FNUM=5.3*pow(10,9)*PRESSURE;
    RECOMB=crec*1.0*pow(10,-13);
    EMOB=QSM/FNUM;
    ETEM=2.0*abs(cene);
    dife=EMOB*ETEM;
    difa=dife/100.0;
    nu2omeg2=pow(OMEG,2)+pow(FNUM,2);
    nu_omeg=FNUM/OMEG;

    //!================Initial density location =======================

    for(K=1;K<=nini;K++)
    {
	      imid[K]=0;
	      jmid[K]=0;
        c_imid[K]=0;
	      c_jmid[K]=0;
	   
	      if(xmid[K]>0.0) 
        {
	        
        imid[K] = xpos*nx;
        
        c_imid[K] = xpos*(factor*(nx));
        
                         printf("xmid[%d]=%f\n",K,xmid[K] ); 
                         printf("imid[%d]=%d\n",K,imid[K] );
                         printf("c_imid[%d]=%d\n",K,c_imid[K] );
	      } 
	      if(ymid[K]>0.0) 
        {  
	        
          jmid[K] = ypos*ny;
	         
          c_jmid[K] = ypos*(factor*ny);
                  
                 printf("ymid[%d]=%f\n",K,ymid[K] );
                 printf("jmid[%d]=%d\n",K,jmid[K] );
                 printf("c_jmid[%d]=%d\n",K,c_jmid[K] );
	      } 
	   if(xmid[K]==0.0) 
	   {     imid[K] = nx/2.0;
	        c_imid[K] = factor/2.0*nx;
	    }
      if(ymid[K]==0.0) 
	     {
          jmid[K] =ny/2.0;
	        c_jmid[K] = factor/2.0*ny;
      }
    }

    //!=============================================================
    TEMP0=300.0;
    DENG0=PRESSURE/760.0*101300.0/akb/TEMP0;

    radius = nx/5*ds;
    printf("radius: %f\n", radius);
    printf("ds: %f\t c_ds: %f\n", ds,c_ds);
    int i;
    
   

	for (i = 0; i <= root_elec->m; i+=1)
	{
		for (j = 0; j <= root_elec->n; j+=1)
		{
			root_den->mesh[i][j] = 0.0;
			root_elec->mesh[i][j] = E0/sqrt(2.0);
		}
	}

	for (i = 0; i <= child_elec->m; i+=1)
	{
		for (j = 0; j <=child_elec->n; j+=1)
		{
			child_den->mesh[i][j] = 0;
			child_elec->mesh[i][j] = E0/sqrt(2.0);
		}
	}


    for(K=1;K<=nini;K++)
    {
	    xd0=ds*imid[K];
        // printf("%.20f\t%.20f\t%d\n",xd0,ds,imid[K]);

	    yd0=ds*jmid[K];
	    c_xd0=c_ds*c_imid[K];
	    c_yd0=c_ds*c_jmid[K];
	    //printf("%f %f\n",xd0,yd0 );
	    if(xmid[K]<0) 
	        xd0=-xmid[K];
	    if(ymid[K]<0) 
	        yd0=-ymid[K];

	    if(xmid[K]<0) 
	        c_xd0=-xmid[K];
	    if(ymid[K]<0) 
	        c_yd0=-ymid[K];

	    //!================Initial density, Gaussian, defined =======================
	    //!make DEN and DENP =0
	    int j;
        
	    for(i=0;i<=nx;i++)
	    {
	        xxi=ds*i;
	        ardix=0.0;
	        if(sgdx0[K]>0)
	            ardix=(-pow((xxi-xd0),2))/2.0/sgdx0[K]/sgdx0[K];

            for(j=0;j<=ny;j++){    
                yyj=ds*j;
                ardiy=0.0;
                if(sgdy0[K]>0) 
                   ardiy=-pow((yyj-yd0),2)/2.0/sgdy0[K]/sgdy0[K];
                dinig=DINI[K]*exp(ardix+ardiy);
                //if(dinig<=DINI[K]*1.0*exp(-2)) 
                  if(dinig<=1.0e13)
                    dinig=0;
                // den[i][j]=den[i][j]+dinig;
                root_den->mesh[i][j] = root_den->mesh[i][j] + dinig;

            }
	    }
		int ic=xpos*nx,jc=ypos*ny;
		printf("root_den->mesh[%d][%d]=%e \n",ic,jc, root_den->mesh[ic][jc]);
			
	    // exit(0);

	    for(i=0;i<=child_den->m;i+=1)
	    {
	    	c_xxi=c_ds*(i+factor*child_den->locx);
	    	c_ardix=0.0;
	    	if(sgdx0[K]>0)
	            c_ardix=(-pow((c_xxi-c_xd0),2))/2.0/sgdx0[K]/sgdx0[K];
	    	for(j=0;j<=child_den->n;j+=1)
	    	{
	    		// printf("%d\n",child_den->n );
	    		c_yyj=c_ds*(j+factor*child_den->locy);
                c_ardiy=0.0;
                if(sgdy0[K]>0){
                	// printf("Happens!!\n");
                   c_ardiy=-pow((c_yyj-c_yd0),2)/2.0/sgdy0[K]/sgdy0[K];
                }
                dinig=DINI[K]*exp(c_ardix+c_ardiy);
                //if(dinig<=DINI[K]*1.0*exp(-2)) 
                  if(dinig<=1.0e13)
                    dinig=0;
                child_den->mesh[i][j] = child_den->mesh[i][j] + dinig;
	    	}
	    }
    }
    dte=dt/eps0;
    dtm=dt/xmu0;
    dteds=dte/ds;
    dtmds=dtm/ds;

    c_dte=c_dt/(eps0);
    c_dtm=c_dt/(xmu0);
    c_dteds=c_dte/c_ds;
    c_dtmds=c_dtm/c_ds;
  
    //mur constants
    c1=(c*dt-ds)/(c*dt+ds);
    c2=2.0*ds/(c*dt+ds);
    c3=(c*dt*c*dt)/(2.0*ds*(c*dt+ds));

    c_c1 = (c*c_dt-c_ds)/(c*c_dt+c_ds);
    
           panim();
           panimE();
           canim();
		       canimE();
    
    
}

void SETUP2()
{

    static int k,contr=-1,contr2=-1;
    static double ardiy,ardix,dinig;
    static double xd0,yd0,xxi,yyj;
    static double c_ardiy,c_ardix,c_dinig;
    static double c_xd0,c_yd0,c_xxi,c_yyj;
    
    static int xtotal2,ytotal2;
    

  if(n==0){
     
   //!   The code should be run for factor =2 /4 not more refinement for coarse nlamb= 256/128 minimum for accuracy,
  //! for  faster experiments to check whether parallelization working , nlamb=64 can be used not below, use E0=5e6 instead of 5.5e6
  //!  to properly resolve the field and density
  //! Code only works for power of 2 (refinement factor only 2,4) and nlamb = 64, 128, 256 ... (parent ) child (512 min for accuracy)
  if(nlamb<64){
  printf("minimum cells required for accuracy is nlamb=128 factor=4 use nlamb=64 for small experiments\n");
  exit(0);
  }
   
     	
          //!------------Activate for MR ---------------- //! for static mesh refinement (MR)
   if(option==0)
   { xend=(xe*nx);yend=(ye*ny);xstar=(xs*nx);ystar=(ys*ny);}
  
  
   //!--------------------Activate for dynamic MR --------------- 
   if(option==1)
   {
      //! The expanding box grows towards left in x, top and bottom along y, 
      //!the right side expansion along x is restricted as no filament propagation

       inibxs= (int)(bxsize*nlamb);  //!inibxs: initial box size x
       inibys= (int)(bysize*nlamb);  //!inibys: initial box size y
       
       xstar=(int)(xpos*nx)-(int)(inibxs*0.5); //!inibxs: initial box size x, xstar: start cell of the child mesh location on parent 
       
       xend= (int)(xpos*nx)+(int)(inibxs*0.5); //!inibxs: initial box size x  xend: end cell of the child mesh location on parent 
       
       ystar=(int)(ypos*ny)-(int)(inibys*0.5); //!inibys: initial box size y  ystar: start cell of the child mesh location on parent 
       
       yend= (int)(ypos*ny)+(int)(inibys*0.5); //!inibys: initial box size y  yend: end cell of the child mesh location on parent
       
       xstarfix=xstar;    //! use for calculating fixed cells to increament along  x (towards left : source side)
       ystarfix=ystar;    //! use for calculating fixed cells to increament along y (towards bottom)
       yendfix=yend;      //! use for calculating fixed cells to increament along y (towards top)
       xstarol=xstar;   //! previous start cell number of initial box along x (left)
       ystarol=ystar;   //! previous start cell number of initial box along y (bottom)
       yendol=yend;     //! previous start cell number of initial box along y  (top)
   
       xinc=(int)(incrx*xstarfix);
       yincs=(int)(incry*ystarfix);
       //yincn=(int)(incry*yendfix);
       
       printf("initial size of box x =%d\n",inibxs);
       printf("initial size of box y =%d\n",inibys);
       printf("Before,xend=%d\tyend=%d\txstar=%d\tystar=%d\t\n",xend,yend,xstar,ystar);
       printf("xstarol=%d ystarol=%d yendol=%d\n",xstarol,ystarol,yendol); 
       printf("xstarfix=%d ystarfix=%d yendfix=%d\n",xstarfix,ystarfix,yendfix); 
       printf("No.expboxcells xs = %d ys = %d yen = %d\n",xinc,yincs,yincs);
       //!----------------------------------------------------------------------- 
    
    //! The cells calculation along x and y for two conditions (a: minimum 25 cells required x and y (each side) expansion)
    //!                       (b: minimum nx>=128 (coarse and fine (factor =4) :512 min. cells per lambda accuracy))
    
    if(xinc>=30){
     xinc=(int)(incrx*xstarfix); 
     //xinc=40;  //activate if bigger domain required
     }
     else{
    
        if(nx>=128)
         {xinc=30;}
         else
         {xinc=20;}
    }
     
     if(yincs>=30){
     yincs=(int)(incry*ystarfix);
     //yincs=40;    //!activate if bigger domain required
     }
     else{
     
     if(ny>=128)
       {yincs=30;}
     else
       {yincs=20;}
     }
    
    }
    xtotal2=(xend-xstar)*factor;      //! Total cells in the refine mesh x-direction
    ytotal2=(yend-ystar)*factor;      //! Total cells in the refine mesh y-direction
    
    printf("after,xend=%d\tyend=%d\txstar=%d\tystar=%d\t\n",xend,yend,xstar,ystar);
//!--------------------------------------------------------------
	
		//!========== Mesh generation (initial) =====================

    //!---------E-field----------
    root_elec = newnode(nx, ny, 0, 0, 0);
	
    child_elec = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  	
  	root_elec->mesh = root_mesh_elec;
  	child_elec->mesh = child_mesh_elec;
  
  	root_elec->children[0] = child_elec;
  	root_elec->parent = NULL;
  
  	child_elec->parent = root_elec;

   //!---------H-field----------
  	root_mag = newnode(nx, ny, 0, 0, 0);
  	child_mag = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  
  	root_mag->mesh = root_mesh_mag;
  	child_mag->mesh = child_mesh_mag;
  
  	root_mag->children[0] = child_mag;
  	root_mag->parent = NULL;
  
  	child_mag->parent = root_mag;

     //!---------electron density----------
  	root_den = newnode(nx, ny, 0, 0, 0);
  	child_den = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  
  	root_den->mesh = root_mesh_den;
  	child_den->mesh = child_mesh_den;
  
  	root_den->children[0] = child_den;
  	root_den->parent = NULL;
  
  	child_den->parent = root_den;
   
  //! ============== parent and child time and grid step (space), constants=======================================
    dt=1.0/(double)(nperdt)/FREQ;
    ds=c/(double)(nlamb)/FREQ;

    c_dt=dt/factor;
    c_ds=ds/factor;

    OMEG=2.0*pi*FREQ;
    QSM=qe/cmasse;
    e2_epsm=qe*qe/eps0/cmasse;
   
   
    FNUM=5.3*pow(10,9)*PRESSURE;
    RECOMB=crec*1.0*pow(10,-13);
    EMOB=QSM/FNUM;
    ETEM=2.0*abs(cene);
    dife=EMOB*ETEM;
    difa=dife/100.0;
    nu2omeg2=pow(OMEG,2)+pow(FNUM,2);
    nu_omeg=FNUM/OMEG;
    
    TEMP0=300.0;
    DENG0=PRESSURE/760.0*101300.0/akb/TEMP0;

    // int i;
     
    //  for (i = 0; i <= root_elec->m; i+=1)
  	// {
  	// 	for (j = 0; j <= root_elec->n; j+=1)
  	// 	{
  	// 		root_den->mesh[i][j] = 0.0;
  	// 		root_elec->mesh[i][j] = E0/sqrt(2.0);
  	// 	}
  	// }
  
  	// for (i = 0; i <= child_elec->m; i+=1)
  	// {
  	// 	for (j = 0; j <=child_elec->n; j+=1)
  	// 	{
  	// 		child_den->mesh[i][j] = 0;
  	// 		child_elec->mesh[i][j] = E0/sqrt(2.0);
  	// 	}
  	// }
      hipMalloc((void**)&dev_root_elec, sizeof(root_elec));
      hipMalloc((void**)&dev_den, sizeof(root_den));
      hipMalloc((void**)&dev_E0, sizeof(double));
      hipMemcpy(dev_root_elec, root_elec, sizeof(root_elec), hipMemcpyHostToDevice);
      hipMemcpy(dev_E0, &E0, sizeof(double), hipMemcpyHostToDevice);
      setup_init<<<(ceil(root_elec->m/32),ceil(root_elec->n/32)),(32,32)>>>(dev_root_elec,dev_den,dev_E0);    
    hipMemcpy(root_den, dev_den, sizeof(dev_den), hipMemcpyDeviceToHost);
    hipMemcpy(root_elec, dev_root_elec, sizeof(dev_den), hipMemcpyDeviceToHost);
    hipFree(dev_root_elec);
    hipFree(dev_den);
    hipFree(dev_E0);

    hipMalloc((void**)&dev_root_elec, sizeof(child_elec));
      hipMalloc((void**)&dev_den, sizeof(child_den));
      hipMalloc((void**)&dev_E0, sizeof(double));
      hipMemcpy(dev_root_elec, child_elec, sizeof(child_elec), hipMemcpyHostToDevice);
      hipMemcpy(dev_E0, &E0, sizeof(double), hipMemcpyHostToDevice);
      setup_init<<<(ceil(child_elec->m/32),ceil(child_elec->n/32)),(32,32)>>>(dev_root_elec,dev_den,dev_E0);    
    hipMemcpy(child_den, dev_den, sizeof(dev_den), hipMemcpyDeviceToHost);
    hipMemcpy(child_elec, dev_root_elec, sizeof(dev_den), hipMemcpyDeviceToHost);
    hipFree(dev_root_elec);
    hipFree(dev_den);
    hipFree(dev_E0);

            
      //!========================= centering the quantities in the new mesh refinement region =============
      
      //!================Initial density location =======================

    for(K=1;K<=nini;K++)
    {
	      imid[K]=0;
	      jmid[K]=0;
        c_imid[K]=0;
	      c_jmid[K]=0;
	   
	      if(xmid[K]>0.0) 
        {
	        
        imid[K] = xpos*(nx);
        
        c_imid[K] = xpos*(factor*nx);         
                        
	      } 
	      if(ymid[K]>0.0) 
        {  
	        
          jmid[K] = ypos*ny;
	         
          c_jmid[K] = ypos*(factor*ny);          
                 
	      } 
    
    }

    //!=============================================================
          
       for(K=1;K<=nini;K++)
    {
	    xd0=ds*imid[K];
      yd0=ds*jmid[K];
	    c_xd0=c_ds*c_imid[K];
	    c_yd0=c_ds*c_jmid[K];
	    
      
	    //!================Initial density, Gaussian, defined =======================
	    
	   // int j;

	    // for(i=0;i<=nx;i++)
	    // {
	    //     xxi=ds*i;
	    //     ardix=0.0;
	    //     if(sgdx0[1]>0)
	    //        ardix=(-pow((xxi-xd0),2))/2.0/sgdx0[1]/sgdx0[1];

        //     for(j=0;j<=ny;j++){    
        //         yyj=ds*j;
        //         ardiy=0.0;
        //         if(sgdy0[K]>0) 
        //           ardiy=-pow((yyj-yd0),2)/2.0/sgdy0[K]/sgdy0[K];
        //          dinig=DINI[K]*exp(ardix+ardiy);
        //          if(dinig<=1.0e13)
        //            dinig=0;
                 
        //         root_den->mesh[i][j] = root_den->mesh[i][j]+ dinig;
        //         denp[i][j]=root_den->mesh[i][j];
        //     }
	    // }

        

        hipMalloc((void**)&dev_den, sizeof(root_den));
        hipMalloc((void**)&dev_ny, sizeof(int));
        hipMalloc((void**)&dev_nx, sizeof(int));
        hipMalloc((void**)&dev_xxi, sizeof(double));
        hipMalloc((void**)&dev_ds, sizeof(double));
        hipMalloc((void**)&dev_ardix, sizeof(double));
        hipMalloc((void**)&dev_yyj, sizeof(double));
        hipMalloc((void**)&dev_ardiy, sizeof(double));
        hipMalloc((void**)&dev_yd0, sizeof(double));
        hipMalloc((void**)&dev_dinig, sizeof(double));
        hipMalloc((void**)&dev_sgdx0, sizeof(sgdx0));
        hipMalloc((void**)&dev_sgdy0, sizeof(dev_sgdy0));
        hipMalloc((void**)&dev_DINI, sizeof(dev_DINI));
        hipMalloc((void**)&dev_K, sizeof(K));

        hipMemcpy(root_den, dev_den, sizeof(dev_den), hipMemcpyDeviceToHost);
        hipMemcpy(dev_ny, &ny, sizeof(ny), hipMemcpyDeviceToHost);
        hipMemcpy(dev_nx, &nx, sizeof(nx), hipMemcpyDeviceToHost);
        hipMemcpy(dev_xxi, &xxi, sizeof(xxi), hipMemcpyDeviceToHost);
        hipMemcpy(dev_ds, &ds, sizeof(ds), hipMemcpyDeviceToHost);
        hipMemcpy(dev_ardix, &ardix, sizeof(ardix), hipMemcpyDeviceToHost);
        hipMemcpy(dev_yyj, &yyj, sizeof(yyj), hipMemcpyDeviceToHost);
        hipMemcpy(dev_ardiy, &ardiy, sizeof(ardiy), hipMemcpyDeviceToHost);
        hipMemcpy(dev_xd0, &xd0, sizeof(xd0), hipMemcpyDeviceToHost);
        hipMemcpy(dev_yd0, &yd0, sizeof(yd0), hipMemcpyDeviceToHost);
        hipMemcpy(dev_dinig, &dinig, sizeof(dinig), hipMemcpyDeviceToHost);
        hipMemcpy(dev_sgdx0, sgdx0, sizeof(sgdx0), hipMemcpyDeviceToHost);
        hipMemcpy(dev_sgdy0, sgdy0, sizeof(sgdy0), hipMemcpyDeviceToHost);
        hipMemcpy(dev_DINI, DINI, sizeof(DINI), hipMemcpyDeviceToHost);
        hipMemcpy(dev_K, &K, sizeof(K), hipMemcpyDeviceToHost);

        
        setup_init1<<<(ceil(nx/32),ceil(ny/32)),(32,32)>>>(dev_den, dev_ny, dev_nx, dev_xxi, dev_ds, dev_ardix,dev_yyj,dev_ardiy,dev_xd0,dev_yd0,dev_dinig,dev_sgdx0,dev_sgdy0,dev_DINI,dev_K);

        hipMemcpy(root_den, dev_den, sizeof(dev_den), hipMemcpyDeviceToHost);
        hipMemcpy(denp, dev_den->mesh, sizeof(dev_den->mesh), hipMemcpyDeviceToHost);

        hipFree(dev_den);
        hipFree(dev_ny);
        hipFree(dev_nx);
        hipFree(dev_xxi);
        hipFree(dev_ds);
        hipFree(dev_ardix);
        hipFree(dev_yyj);
        hipFree(dev_ardiy);
        hipFree(dev_xd0);
        hipFree(dev_yd0);
        hipFree(dev_dinig);
        hipFree(dev_sgdx0);
        hipFree(dev_sgdy0);
        hipFree(dev_DINI);
  
          printf("Entered here \n");
          interpolatexinitial(1.0);    //! to interpolate the initial gaussian density parent to child location
          printf("Entered here \n");
          
          printf("left here \n");
    }
      
    
    dte=dt/eps0;
    dtm=dt/xmu0;
    dteds=dte/ds;
    dtmds=dtm/ds;

    c_dte=c_dt/(eps0);
    c_dtm=c_dt/(xmu0);
    c_dteds=c_dte/c_ds;
    c_dtmds=c_dtm/c_ds;
  
    //mur constants
    c1=(c*dt-ds)/(c*dt+ds);
    c2=2.0*ds/(c*dt+ds);
    c3=(c*dt*c*dt)/(2.0*ds*(c*dt+ds));

    c_c1 = (c*c_dt-c_ds)/(c*c_dt+c_ds);
    
    //!======== Initialize the child electric field , velcocity, density and magnetic field in the new refinement region ========

          
  
    //!=============================================================
    
      
           printf("Entered here \n");
           panim();
           panimE();
           canim();
		       canimE(); 
 
  }
 if(n>0){  
     
    if(option==1){
    if(chc==1)
   {
      //! cells expansion in x , must not go below the iend or xs*nx
     if(xstarol>xinc)
     {
     xstarnew=xstarol-xinc; 
    
     if(xstarnew>=(int)(xs*nx))
     { xstar=xstarnew;}
     else
      {xstar=(int)(xs*nx);}
     
     }  
     else
     {
       xstar=(int)(xs*nx);
       xstarnew=xstar;
     }
   
   }
   
   if(chc==2)
   {
      //! cells expansion in y , must not go below the ystar (final) or ys*ny
     if(ystarol>yincs)
     {
       ystarnew=ystarol-yincs; 
       
       if(ystarnew>=(int)(ys*ny))
       { ystar=ystarnew;}
       else
       { ystar=(int)(ys*ny);} 
     }
     else
     {
        ystar=(int)(ys*ny);
        ystarnew=ystar;
     }
   }
  
   if(chc==2)
   {
      //! cells expansion in y , must not go above the ystar (final) or ye*ny
     if(yendol<=(int)(ye*ny))
     {
       yendnew=yendol+yincs; 
       
       if(yendnew<=(int)(ye*ny))
        {yend=yendnew;}
       else
        {yend=(int)(ye*ny);}
     }
     else
     {
        yend=(int)(ye*ny);
        yendnew=yend;
     }
   
   }
   
    xtotal2=(xend-xstar)*factor;
    ytotal2=(yend-ystar)*factor;
    printf("after,xend=%d\tyend=%d\txstar=%d\tystar=%d\t\n",xend,yend,xstar,ystar);
//!--------------------------------------------------------------
	
	 //!========== Mesh generation (expanding) =====================
    //! every steps follows same technique: mesh generation, define the parent and child relation , here only one child 
    //! no further division exists
    //!---------E-field----------	
    root_elec = newnode(nx, ny, 0, 0, 0);
	
    child_elec = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  	
  	root_elec->mesh = root_mesh_elec;
  	child_elec->mesh = child_mesh_elec;
  
  	root_elec->children[0] = child_elec;
  	root_elec->parent = NULL;
  
  	child_elec->parent = root_elec;

    //!---------H-field---------- 
  	root_mag = newnode(nx, ny, 0, 0, 0);
  	child_mag = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  
  	root_mag->mesh = root_mesh_mag;
  	child_mag->mesh = child_mesh_mag;
  
  	root_mag->children[0] = child_mag;
  	root_mag->parent = NULL;
  
  	child_mag->parent = root_mag;

    //!---------electron density---------- 
  	root_den = newnode(nx, ny, 0, 0, 0);
  	child_den = newnode(xtotal2, ytotal2, xstar, ystar, 1);
  
  	root_den->mesh = root_mesh_den;
  	child_den->mesh = child_mesh_den;
  
  	root_den->children[0] = child_den;
  	root_den->parent = NULL;
  
  	child_den->parent = root_den;
  
   //!=======================================
    printf("Entered here \n");
    //! chc==1/2 indicate the interpolation correspond to x or y expansion
    if(chc==1)
    {
      interpolatexnew(1.0);   //! interpolates all previous mesh data along x
      printf("Entered here \n");
    }
    if(chc==2)
    {
      interpolateynew(1.0);   //! interpolates all previous mesh data along y
      printf("Entered here \n");
    }
    
    printf("left here \n");

    if(chc==1)
   {
       if(xstarnew>(int)(xs*nx))
       xstarol=xstarnew;
       
   }
   
   if(chc==2)
   {
        if(ystarnew>(int)(ys*ny))
        { ystarol=ystarnew;}
        
        if(yendnew<(int)(ye*ny))
        { yendol=yendnew;}
        
    } 
     
    
  }
 
 }                  

}


//! to perform interpolation of parent data f(x,y): f represents either E-,H-, electron density etc., on child at 2d positions 
//! bilinear interpolation
double interpolate2d(double a,double b,double c,double d, double posx, double posy)
{
	double val;
	val = (1-posx)*(1-posy)*a + (posx)*(1-posy)*b + (posx)*posy*c + (1-posx)*posy*d;
	return val;
}

//! to perform interpolation of parent data f(x) or f(y): f represents either E-,H-, electron density etc., on child at 1d positions 
//! linear interpolation
double interpolate1d(double a, double b, double posx)
{
	return ((1-posx)*a + b*posx);
}

//! perform interpolation at the corners (density), at center (H-field), edge (E-field) only on the cpb (child parent boundary)
void interpolatecorners(double t1)
{
	double tau = t1;
	int basex, basey;
	double distx, disty;
	//int factor=8;
	if(child_elec->locx>0)
	{
		basex = child_elec->locx-1;
		basey = child_elec->locy;
		distx =	0.5+ 0.5/factor;
    	c_exs[0][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],
			exs_old[basex+1][basey],distx))+(tau)*(interpolate1d(
			exs[basex][basey],exs[basex+1][basey],distx));
    					
    	c_vx[0][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],
		     vx_old[basex+1][basey],distx))+(tau)*(interpolate1d(
		     vx[basex][basey],vx[basex+1][basey],distx));
	}

	if(child_elec->locy>0)
	{
		basex = child_elec->locx;
		basey = child_elec->locy-1;
		disty = 0.5+0.5/factor;
		c_eys[0][0] = (1-tau)*interpolate1d(eys_old[basex][basey],
			       eys_old[basex][basey+1],disty) + 
			      (tau)*interpolate1d(eys[basex][basey],
			       eys[basex][basey+1],disty);  
		c_vy[0][0] = (1-tau)*interpolate1d(vy_old[basex][basey],
			     vy_old[basex][basey+1],disty) + (tau)*interpolate1d(			      vy[basex][basey],vy[basex][basey+1],disty);
	}

	if(child_elec->locx>0&&child_elec->locy>0)
	{
		basex = child_elec->locx-1;
		basey = child_elec->locy-1;
		distx = 0.5 + 0.5/factor;
		disty = 0.5 + 0.5/factor;
		child_mag->mesh[0][0] = (1-tau)*(interpolate2d(hzi[basex][basey],					  hzi[basex+1][basey],
					  hzi[basex+1][basey+1],
					 hzi[basex][basey+1],distx,disty))+
					(tau)*(interpolate2d(
					 root_mag->mesh[basex][basey],
					root_mag->mesh[basex+1][basey],
					root_mag->mesh[basex+1][basey+1],
					root_mag->mesh[basex][basey+1],
					distx,disty));
	}

	child_den->mesh[0][0] = (1-tau)*denp[child_den->locx][child_den->locy] + 			     tau*root_den->mesh[child_den->locx][child_den->locy];

	basex = child_elec->locx+(child_elec->m)/factor;
	basey = child_elec->locy-1;
	disty = 0.5 + 0.5/factor;
	c_eys[child_elec->m][0] = (1-tau)*interpolate1d(eys_old[basex][basey],
				  eys_old[basex][basey+1],disty) + 
				  tau*interpolate1d(eys[basex][basey],
				  eys[basex][basey+1],disty);

	basex = child_elec->locx-1;
	basey = child_elec->locy+(child_elec->n)/factor;
	distx = 0.5 + 0.5/factor;
	c_exs[0][child_elec->n] = (1-tau)*interpolate1d(exs_old[basex][basey],
				exs_old[basex+1][basey],distx) + 
				tau*interpolate1d(exs[basex][basey],
				exs[basex+1][basey],distx);

	c_eyt[0][0] = c_eys[0][0] + c_eyi1[0][0];
	c_ext[0][0] = c_exs[0][0] + c_exi1[0][0];

c_ext[0][child_elec->n]= c_exs[0][child_elec->n] + c_exi1[0][child_elec->n];
c_eyt[child_elec->m][0] = c_exs[child_elec->m][0] + c_exi1[child_elec->m][0];

}

void interpolatecornersnew(double t1)
{
	double tau = t1;
	int basex, basey,lx,ly;
	double distx, disty;
	
	if(child_elec->locx>0)
	{
		basex = child_elec->locx-1;
		basey = child_elec->locy;
		distx =	0.5+ 0.5/factor;
    	c_exs[0][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],
			exs_old[basex+1][basey],distx))+(tau)*(interpolate1d(
			exs[basex][basey],exs[basex+1][basey],distx));
    					
    	c_vx[0][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],
		     vx_old[basex+1][basey],distx))+(tau)*(interpolate1d(
		     vx[basex][basey],vx[basex+1][basey],distx));
	}

	if(child_elec->locy>0)
	{
		basex = child_elec->locx;
		basey = child_elec->locy-1;
		disty = 0.5+0.5/factor;
		c_eys[0][0] = (1-tau)*interpolate1d(eys_old[basex][basey],
			       eys_old[basex][basey+1],disty) + 
			      (tau)*interpolate1d(eys[basex][basey],
			       eys[basex][basey+1],disty);  
		c_vy[0][0] = (1-tau)*interpolate1d(vy_old[basex][basey],
			     vy_old[basex][basey+1],disty) + (tau)*interpolate1d(			      vy[basex][basey],vy[basex][basey+1],disty);
	}

	if(child_elec->locx>0&&child_elec->locy>0)
	{
		basex = child_elec->locx-1;
		basey = child_elec->locy-1;
		distx = 0.5 + 0.5/factor;
		disty = 0.5 + 0.5/factor;
		child_mag->mesh[0][0] = (1-tau)*(interpolate2d(hzi[basex][basey],					  hzi[basex+1][basey],
					  hzi[basex+1][basey+1],
					 hzi[basex][basey+1],distx,disty))+
					(tau)*(interpolate2d(
					 root_mag->mesh[basex][basey],
					root_mag->mesh[basex+1][basey],
					root_mag->mesh[basex+1][basey+1],
					root_mag->mesh[basex][basey+1],
					distx,disty));
	
  }

  i=(child_den->m);
  j=(child_den->n);
	child_den->mesh[0][0] = (1-tau)*denp[child_den->locx][child_den->locy] + tau*root_den->mesh[child_den->locx][child_den->locy];
  
  child_den->mesh[0][j] = (1-tau)*denp[child_den->locx][child_den->locy+j/factor] + tau*root_den->mesh[child_den->locx][child_den->locy+j/factor];
  
  child_den->mesh[i][j] = (1-tau)*denp[child_den->locx+i/factor][child_den->locy+j/factor] + tau*root_den->mesh[child_den->locx+i/factor][child_den->locy+j/factor];
  
  child_den->mesh[i][0] = (1-tau)*denp[child_den->locx+i/factor][child_den->locy] + tau*root_den->mesh[child_den->locx+i/factor][child_den->locy];
  
 
	basex = child_elec->locx+(child_elec->m)/factor;
	basey = child_elec->locy-1;
	disty = 0.5 + 0.5/factor;
	c_eys[child_elec->m][0] = (1-tau)*interpolate1d(eys_old[basex][basey],
				  eys_old[basex][basey+1],disty) + 
				  tau*interpolate1d(eys[basex][basey],
				  eys[basex][basey+1],disty);
  

	basex = child_elec->locx-1;
	basey = child_elec->locy+(child_elec->n)/factor;
	distx = 0.5 + 0.5/factor;
	c_exs[0][child_elec->n] = (1-tau)*interpolate1d(exs_old[basex][basey],
				exs_old[basex+1][basey],distx) + 
				tau*interpolate1d(exs[basex][basey],
				exs[basex+1][basey],distx);

	c_eyt[0][0] = c_eys[0][0] + c_eyi1[0][0];
	c_ext[0][0] = c_exs[0][0] + c_exi1[0][0];

c_ext[0][child_elec->n]= c_exs[0][child_elec->n] + c_exi1[0][child_elec->n];
c_eyt[child_elec->m][0] = c_eys[child_elec->m][0] + c_eyi1[child_elec->m][0];

}

void interpolatex(double t1)
{
    gettimeofday(&begin,NULL);
    double tau = t1;
    int basex, basey;
    double distx, disty;
    //int factor=8;
	// printf("interpolatex\n");
/*********************interpolation******************************/
for(j=1;j<child_elec->n;j++)
{

    double ita,ep;
    

    	if(child_elec->locx>0)
    	{
    		basex = child_elec->locx-1;
    		basey = child_elec->locy+j/factor;
		distx = 0.5 + 0.5/factor;
		disty = (double)(j%factor)/factor;
    		
	    	c_exs[0][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
			      exs_old[basex][basey+1],distx,disty))+(tau)*(
			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
			      disty));
	    					
	    	c_vx[0][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
			     vx_old[basex][basey+1],distx,disty))+(tau)*(
			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
			     disty));
    	}

    	if(child_elec->locx+(child_elec->m-1)/factor+1<root_elec->m)
    	{
			basex = child_elec->locx+(child_elec->m-1)/factor;
			basey = child_elec->locy+j/factor;
			distx = 0.5 - 0.5/factor;
			disty = (double)(j%factor)/factor;
			c_exs[child_elec->m-1][j] = (1-tau)*(interpolate2d(
						    exs_old[basex][basey],
						    exs_old[basex+1][basey],
						    exs_old[basex+1][basey+1],
						    exs_old[basex][basey+1],
						    distx,disty))+(tau)*(
						    interpolate2d(
						    exs[basex][basey],
						    exs[basex+1][basey],
					            exs[basex+1][basey+1],
						    exs[basex][basey+1],distx,
						    disty));
		
			c_vx[child_elec->m-1][j] = (1-tau)*(interpolate2d(
						   vx_old[basex][basey],
						   vx_old[basex+1][basey],
						   vx_old[basex+1][basey+1],
						   vx_old[basex][basey+1],distx,
						   disty))+(tau)*(
						   interpolate2d(vx[basex][basey]						    ,vx[basex+1][basey],
						   vx[basex+1][basey+1],
						   vx[basex][basey+1],distx,
						   disty));
		}
    	
	if(j%factor<factor/2)
     {	
		basex = child_elec->locx;
		basey = child_elec->locy+j/factor-1;
		disty = 0.5 + (double)(j%factor)/(factor) + 0.5/(factor);

		c_eys[0][j] = (1-tau)*interpolate1d(eys_old[basex][basey],
			      eys_old[basex][basey+1],disty) + 
			      (tau)*interpolate1d(eys[basex][basey],
			       eys[basex][basey+1],disty);
		c_vy[0][j] = (1-tau)*interpolate1d(vy_old[basex][basey],
			     vy_old[basex][basey+1],disty) + 
			     (tau)*interpolate1d(vy[basex][basey],
			     vy[basex][basey+1],disty);
     }
	else
    {   
 	basex = child_elec->locx;
    	basey = child_elec->locy+j/factor;
	disty = (double)(j%factor)/(factor) + 0.5/(factor)-0.5;

    	c_eys[0][j] = (1-tau)*interpolate1d(eys_old[basex][basey],
		      eys_old[basex][basey+1],disty) + (tau)*interpolate1d(
                      eys[basex][basey],eys[basex][basey+1],disty);
    	c_vy[0][j] = (1-tau)*interpolate1d(vy_old[basex][basey],
		     vy_old[basex][basey+1],disty) + (tau)*interpolate1d(
		     vy[basex][basey],vy[basex][basey+1],disty);
    	
    }	
    
    if (j%factor<factor/2)
    {
		basex = child_elec->locx + (child_elec->m)/factor;
		basey = child_elec->locy+j/factor-1;
		disty = 0.5 + (double)(j%factor)/(factor) + 0.5/(factor);
		
	c_eys[child_elec->m][j] = (1-tau)*(interpolate1d(eys_old[basex][basey],
				  eys_old[basex][basey+1],disty))+(tau)*(
				  interpolate1d(eys[basex][basey],
				  eys[basex][basey+1],disty));

	c_vy[child_elec->m][j] = (1-tau)*(interpolate1d(vy_old[basex][basey],
				vy_old[basex][basey+1],disty))+(tau)*(
				interpolate1d(vy[basex][basey],
				vy[basex][basey+1],disty));
   }
  else
  {
	basex = child_elec->locx + (child_elec->m)/factor;
    	basey = child_elec->locy+j/factor;
	disty = (double)(j%factor)/(factor) + 0.5/(factor)-0.5;

    	c_eys[child_elec->m][j] = (1-tau)*(interpolate1d(eys_old[basex][basey],
				eys_old[basex][basey+1],disty))+(tau)*(
				interpolate1d(eys[basex][basey],
				eys[basex][basey+1],disty));
	
	c_vy[child_elec->m][j] = (1-tau)*(interpolate1d(vy_old[basex][basey],
				vy_old[basex][basey+1],disty))+(tau)*(
				interpolate1d(vy[basex][basey],
				vy[basex][basey+1],disty));

  }

    if(j%factor<factor/2 && child_elec->locx>0)
    {
	basex = child_elec->locx-1;
	basey = child_elec->locy+j/factor-1;
	distx = 0.5 + 0.5/factor;
	disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
	
	child_mag->mesh[0][j] = (1-tau)*(interpolate2d(hzi[basex][basey],
				hzi[basex+1][basey],hzi[basex+1][basey+1],
				hzi[basex][basey+1],distx,disty))+(tau)*(
				interpolate2d(root_mag->mesh[basex][basey],
				root_mag->mesh[basex+1][basey],
				root_mag->mesh[basex+1][basey+1],
				root_mag->mesh[basex][basey+1],distx,disty));
    }

    else if (child_elec->locx>0)
    {
		basex = child_elec->locx-1;
		basey = child_elec->locy+j/factor;
		distx = 0.5 + 0.5/factor;
		disty = 0.5/factor + (double)(j%factor)/factor-0.5;

	child_mag->mesh[0][j] = (1-tau)*(interpolate2d(hzi[basex][basey],
				hzi[basex+1][basey],hzi[basex+1][basey+1],
				hzi[basex][basey+1],distx,disty))+(tau)*(
				interpolate2d(root_mag->mesh[basex][basey],
				root_mag->mesh[basex+1][basey],
				root_mag->mesh[basex+1][basey+1],
				root_mag->mesh[basex][basey+1],distx,disty));
    } 
	
	if(j%factor<factor/2)
        {
                basex = child_elec->locx+(child_elec->m-1)/factor;
                basey = child_elec->locy+j/factor-1;
                distx = 0.5 - 0.5/factor;
                disty = 0.5 + (double)(j%factor)/factor + 0.5/factor;

         child_mag->mesh[child_elec->m-1][j] = (1-tau)*(interpolate2d(
						hzi[basex][basey],
						hzi[basex+1][basey],
						hzi[basex+1][basey+1],
						hzi[basex][basey+1],distx,disty))                                                +(tau)*(interpolate2d(
						root_mag->mesh[basex][basey],
						root_mag->mesh[basex+1][basey],
						root_mag->mesh[basex+1][basey+1],						root_mag->mesh[basex][basey+1],
						distx,disty));
        }

	else
        {
                basex = child_elec->locx+(child_elec->m-1)/factor;
                basey = child_elec->locy+j/factor;
                distx = 0.5 - 0.5/factor;
                disty = (double)(j%factor)/factor + 0.5/factor - 0.5;

         child_mag->mesh[child_elec->m-1][j] = (1-tau)*(interpolate2d(
						hzi[basex][basey],
						hzi[basex+1][basey],
						hzi[basex+1][basey+1],
						hzi[basex][basey+1],distx,disty))
					       +(tau)*(interpolate2d(
						root_mag->mesh[basex][basey],
						root_mag->mesh[basex+1][basey],
						root_mag->mesh[basex+1][basey+1],						 root_mag->mesh[basex][basey+1],
						distx,disty));
        }

	basex = child_elec->locx;
        basey = child_elec->locy+j/factor;
        disty = (double)(j%factor)/factor;
 
        child_den->mesh[0][j] = (1-tau)*(interpolate1d(denp[basex][basey],
				denp[basex][basey+1],disty))+(tau)*(
				interpolate1d(root_den->mesh[basex][basey],
				root_den->mesh[basex][basey+1],disty));

        basex = child_elec->locx+(child_elec->m)/factor;
        basey = child_elec->locy+j/factor;
        disty = (double)(j%factor)/factor;
 
       child_den->mesh[child_elec->m][j] =(1-tau)*(interpolate1d(
					  denp[basex][basey],
					  denp[basex][basey+1],disty))+(tau)*(
					  interpolate1d(
					  root_den->mesh[basex][basey],
					  root_den->mesh[basex][basey+1],
					  disty));

}
    gettimeofday(&end,NULL);
    t_cal_interpolatex += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void interpolatexinitial(double t1)
{
    gettimeofday(&begin,NULL);
    double tau = t1;
    int basex, basey,l,g;
    double distx, disty;
    
/*********************interpolation******************************/
for(i=0;i<=child_elec->m;i++)
{
  for(j=0;j<=child_elec->n;j++)
  {

    double ita,ep;
    double fac=0.25;
       
              //!=========== e-density ============
    
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor;
              distx = (double)(i%factor)/factor;
              disty = (double)(j%factor)/factor;
            
                        
              child_den->mesh[i][j] = (1-tau)*(interpolate2d(denp[basex][basey],
                    denp[basex+1][basey],denp[basex+1][basey+1],
                    denp[basex][basey+1],distx,disty))+(tau)*(
                    interpolate2d(root_den->mesh[basex][basey],
                    root_den->mesh[basex+1][basey],
                    root_den->mesh[basex+1][basey+1],
                    root_den->mesh[basex][basey+1],distx,disty));
                    
                    
                 //!=========== E-field + vel x ============
           
        if((child_elec->locx>0) && (i<child_elec->m))
        {
         if(i%factor<factor/2)
         {
          basex = child_elec->locx+i/factor-1;
          basey = child_elec->locy+j/factor;
          //distx = 0.5 + 0.5/factor;
          distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
          disty = (double)(j%factor)/factor;
          
          c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
              exs_old[basex+1][basey],exs_old[basex+1][basey+1],
              exs_old[basex][basey+1],distx,disty))+(tau)*(
              interpolate2d(exs[basex][basey],exs[basex+1][basey]            ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
              disty));
                  
          c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
             vx_old[basex+1][basey],vx_old[basex+1][basey+1],
             vx_old[basex][basey+1],distx,disty))+(tau)*(
             interpolate2d(vx[basex][basey],vx[basex+1][basey],
             vx[basex+1][basey+1],vx[basex][basey+1],distx,
             disty));
        
               
        }
        else
        {   
          if(child_elec->locx+i/factor+1<(root_elec->m))
          {
          basex = child_elec->locx+i/factor;
          basey = child_elec->locy+j/factor;
          
          distx = 0.5/factor+ (double)(i%factor)/factor - 0.5;
          disty = (double)(j%factor)/factor;
          
          c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
              exs_old[basex+1][basey],exs_old[basex+1][basey+1],
              exs_old[basex][basey+1],distx,disty))+(tau)*(
              interpolate2d(exs[basex][basey],exs[basex+1][basey]            ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
              disty));
                  
          c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
             vx_old[basex+1][basey],vx_old[basex+1][basey+1],
             vx_old[basex][basey+1],distx,disty))+(tau)*(
             interpolate2d(vx[basex][basey],vx[basex+1][basey],
             vx[basex+1][basey+1],vx[basex][basey+1],distx,
             disty));
          }
        }
        
       }
       
        //!=========== E-field + vel y ============
       
     if((child_elec->locy>0) && (j<child_elec->n))
     {
      if(j%factor<factor/2)
      {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor-1;
                distx = (double)(i%factor)/factor;
                
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],
                  eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],
                  vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      }
      
      else
      {
          if(child_elec->locy+j/factor+1<(root_elec->n))
          {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor;
                distx = (double)(i%factor)/factor;
                //disty = 0.5 + 0.5/factor;
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],
                  eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],
                  vy_old[basex+1][basey],vy_old[basex+1][basey+1],
                  vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      
         }
       }
           
      }        
                    
        //!====================== H-field ================
       
       if((child_elec->locy>0) && (j<child_elec->n))
          {
            if(j%factor<factor/2)
            {
              
              if (i%factor<factor/2)
              {  
        
        
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy-1+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                                                                
              }
            
            else
            {
                
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor-1;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
              child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                
             }  
            
            }    
                
       else    
        {
            if (i%factor<factor/2)
             {  
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor; 
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            
              }
            else
            {
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
               child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                 hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                 root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            }
        
          }        
      }       
      
      //!========================================================             
                    
       
    }
}
    gettimeofday(&end,NULL);
    t_cal_interpolatexinitial += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));                 
                    

}

void interpolatexnew(double t1)
{
    gettimeofday(&begin,NULL);
    double tau = t1;
    int basex, basey,l,g;
    double distx, disty;
    
/*********************interpolation along x(expanding box)******************************/
for(i=0;i<=child_elec->m;i++)
{
  for(j=0;j<=child_elec->n;j++)
  {

    double ita,ep;
    double fac=0.25;
       
    
              basex = child_elec->locx+i/factor;
            	basey = child_elec->locy+j/factor;
              distx = (double)(i%factor)/factor;
              disty = (double)(j%factor)/factor;
            
                      	
            	child_den->mesh[i][j] = (1-tau)*(interpolate2d(denp[basex][basey],
            				denp[basex+1][basey],denp[basex+1][basey+1],
            				denp[basex][basey+1],distx,disty))+(tau)*(
            				interpolate2d(root_den->mesh[basex][basey],
            				root_den->mesh[basex+1][basey],
            				root_den->mesh[basex+1][basey+1],
            				root_den->mesh[basex][basey+1],distx,disty));
           
          
            if((child_den->locx+i/factor)>=xstarol && i<=child_elec->m)
               {
                     	child_den->mesh[i][j] = c_denpold[i-(int)((xstarol-xstar)*factor)][j];
                     
               }
            
                   
             if((child_den->locx+i/factor)==xstarol)
             {
                    basex = child_elec->locx+i/factor;
                  	basey = child_elec->locy+j/factor;
                    distx = (double)(i%factor)/factor;
                    disty = (double)(j%factor)/factor;
             
                   	child_den->mesh[i][j] = (1-fac)*(c_denpold[i-(int)((xstarol-xstar)*factor)][j])+(fac)*(
                    interpolate2d(root_den->mesh[basex][basey],
            				root_den->mesh[basex+1][basey],
            				root_den->mesh[basex+1][basey+1],
            				root_den->mesh[basex][basey+1],distx,disty));
                  
             }
           
            //!=========== E-field + vel x ============
           
        if((child_elec->locx>0) && (i<child_elec->m))
        {
         if(i%factor<factor/2)
         {
          basex = child_elec->locx+i/factor-1;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
        
               
        }
        else
        {   
          if(child_elec->locx+i/factor+1<(root_elec->m))
          {
          basex = child_elec->locx+i/factor;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5/factor+ (double)(i%factor)/factor - 0.5;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
          }
        }
        
    	 }
          
         if((child_elec->locx+i/factor)>=xstarol && i<child_elec->m)
               {
                     	c_exs[i][j] = c_exold[i-(int)((xstarol-xstar)*factor)][j];
                      
                      c_vx[i][j] = c_vxold[i-(int)((xstarol-xstar)*factor)][j];
               } 
          
             
       //!=========== E-field + vel y ============
       
     if((child_elec->locy>0) && (j<child_elec->n))
     {
      if(j%factor<factor/2)
      {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor-1;
                distx = (double)(i%factor)/factor;
                
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],
                  eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],
                  vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      }
      
      else
      {
          if(child_elec->locy+j/factor+1<(root_elec->n))
          {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor;
                distx = (double)(i%factor)/factor;
                
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],
                  eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],
                  vy_old[basex+1][basey],vy_old[basex+1][basey+1],
                  vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      
         }
       }
           
      }
         
          if((child_elec->locx+i/factor)>=xstarol && i<child_elec->m)
               {
                     	c_eys[i][j] = c_eyold[i-(int)((xstarol-xstar)*factor)][j];
                      
                      c_vy[i][j] = c_vyold[i-(int)((xstarol-xstar)*factor)][j];
               } 
        
     
       //!====================== H-field ================
       
       if((child_elec->locy>0) && (j<child_elec->n))
          {
            if(j%factor<factor/2)
            {
              
              if (i%factor<factor/2)
              {  
        
        
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy-1+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                                                                
              }
            
            else
            {
                
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor-1;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
              child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                
             }  
            
            }    
                
       else    
        {
            if (i%factor<factor/2)
             {  
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor; 
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            
              }
            else
            {
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
               child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                 hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                 root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            }
        
          }        
      }          
         
         if((child_elec->locx+i/factor)>=xstarol && i<child_elec->m)
               {
                     	child_mag->mesh[i][j] = c_hzold[i-(int)((xstarol-xstar)*factor)][j];
                      
               } 
   
       
      //!===================================     
         
      
    
    }
}
    gettimeofday(&end,NULL);
    t_cal_interpolatexnew += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void interpolateynew(double t1)
{
    gettimeofday(&begin,NULL);
    double tau = t1;
    int basex, basey,l,g;
    double distx, disty;
    
/*********************interpolation along y(expanding box)******************************/
for(i=0;i<=child_elec->m;i++)
{
  for(j=0;j<=child_elec->n;j++)
  {
    double ita,ep;
    double fac=0.25;
       
    
              basex = child_elec->locx+i/factor;
            	basey = child_elec->locy+j/factor;
              distx = (double)(i%factor)/factor;
              disty = (double)(j%factor)/factor;
            
                      	
            	child_den->mesh[i][j] = (1-tau)*(interpolate2d(denp[basex][basey],
            				denp[basex+1][basey],denp[basex+1][basey+1],
            				denp[basex][basey+1],distx,disty))+(tau)*(
            				interpolate2d(root_den->mesh[basex][basey],
            				root_den->mesh[basex+1][basey],
            				root_den->mesh[basex+1][basey+1],
            				root_den->mesh[basex][basey+1],distx,disty));
           
          
              if((child_den->locy+j/factor)>=(ystarol) && (child_den->locy+j/factor)<=(yendol) )
              
               {
                        child_den->mesh[i][j] = c_denpold[i][j-(int)((ystarol-ystar)*factor)];
                    
               }
            
                   
            
             if((child_den->locy+j/factor)==ystarol || (child_den->locy+j/factor)==(yendol))
             {
                    basex = child_elec->locx+i/factor;
                  	basey = child_elec->locy+j/factor;
                    distx = (double)(i%factor)/factor;
                    disty = (double)(j%factor)/factor;
             
                   child_den->mesh[i][j] = (1-fac)*(c_denpold[i][j-(int)((ystarol-ystar)*factor)])+(fac)*(
                    interpolate2d(root_den->mesh[basex][basey],
            				root_den->mesh[basex+1][basey],
            				root_den->mesh[basex+1][basey+1],
            				root_den->mesh[basex][basey+1],distx,disty));
                 
             }
           
            //!=========== E-field + vel x ============
           
        if((child_elec->locx>0) && (i<child_elec->m))
        {
         if(i%factor<factor/2)
         {
          basex = child_elec->locx+i/factor-1;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
        
               
        }
        else
        {   
          if(child_elec->locx+i/factor+1<(root_elec->m))
          {
          basex = child_elec->locx+i/factor;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5/factor+ (double)(i%factor)/factor - 0.5;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
          }
        }
        
    	 }
          
        
              if((child_den->locy+j/factor)>=(ystarol) && (child_den->locy+j/factor)<=(yendol) )
               {
                     	c_exs[i][j] = c_exold[i][j-(int)((ystarol-ystar)*factor)];
                      
                      c_vx[i][j] = c_vxold[i][j-(int)((ystarol-ystar)*factor)];
               } 
          
             
       //!=========== E-field + vel y ============
       
     if((child_elec->locy>0) && (j<child_elec->n))
     {
      if(j%factor<factor/2)
      {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor-1;
                distx = (double)(i%factor)/factor;
                
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],
                  eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],
                  vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      }
      
      else
      {
          if(child_elec->locy+j/factor+1<(root_elec->n))
          {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor;
                distx = (double)(i%factor)/factor;
                
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],
                  eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],
                  eys_old[basex][child_elec->locy],distx,disty))+(tau)*(interpolate2d(eys[basex][basey],
                  eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],
                  vy_old[basex+1][basey],vy_old[basex+1][basey+1],
                  vy_old[basex][basey+1],distx,disty))+(tau)*(interpolate2d(vy[basex][basey],
                  vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      
         }
       }
           
      }
         
         
               if((child_den->locy+j/factor)>=(ystarol) && (child_den->locy+j/factor)<=(yendol) )
               {
                     	c_eys[i][j] = c_eyold[i][j-(int)((ystarol-ystar)*factor)];
                      
                      c_vy[i][j] = c_vyold[i][j-(int)((ystarol-ystar)*factor)];
               } 
        
     
       //!====================== H-field ================
       
       if((child_elec->locy>0) && (j<child_elec->n))
          {
            if(j%factor<factor/2)
            {
              
              if (i%factor<factor/2)
              {  
        
        
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy-1+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                                                                
              }
            
            else
            {
                
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor-1;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
              child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                
             }  
            
            }    
                
       else    
        {
            if (i%factor<factor/2)
             {  
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor; 
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                  hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                  root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            
              }
            else
            {
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
               child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],
                 hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))+(tau)*(interpolate2d(root_mag->mesh[basex][basey],
                 root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            }
        
          }        
      }          
         
        
               if((child_den->locy+j/factor)>=(ystarol) && (child_den->locy+j/factor)<=(yendol) ) 
               {
                     	child_mag->mesh[i][j] = c_hzold[i][j-(int)((ystarol-ystar)*factor)];
                      
               } 
   
       
      //!===================================     
       
     
   
       
    }
}
    gettimeofday(&end,NULL);
    t_cal_interpolateynew += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void interpolatexall(double t1)
{
    gettimeofday(&begin,NULL);
    double tau = t1;
    int basex, basey;
    double distx, disty;
    
/*********************interpolation******************************/
for(i=0;i<=child_elec->m;i++)
{
  for(j=1;j<child_elec->n;j++)
  {
//	int factor=2;
    double ita,ep;

 
    
    if((child_elec->locx>0) && (i<child_elec->m))
    {
       if(i%factor<factor/2)
      {
          basex = child_elec->locx+i/factor-1;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
        }
      else
      {   
          if(child_elec->locx+i/factor+1<(root_elec->m))
          {
          basex = child_elec->locx+i/factor;
      		basey = child_elec->locy+j/factor;
      		
          distx = 0.5/factor+ (double)(i%factor)/factor - 0.5;
      		disty = (double)(j%factor)/factor;
      		
  	    	c_exs[i][j] = (1-tau)*(interpolate2d(exs_old[basex][basey],
  			      exs_old[basex+1][basey],exs_old[basex+1][basey+1],
  			      exs_old[basex][basey+1],distx,disty))+(tau)*(
  			      interpolate2d(exs[basex][basey],exs[basex+1][basey]			       ,exs[basex+1][basey+1],exs[basex][basey+1],distx,
  			      disty));
  	    					
  	    	c_vx[i][j] = (1-tau)*(interpolate2d(vx_old[basex][basey],
  			     vx_old[basex+1][basey],vx_old[basex+1][basey+1],
  			     vx_old[basex][basey+1],distx,disty))+(tau)*(
  			     interpolate2d(vx[basex][basey],vx[basex+1][basey],
  			     vx[basex+1][basey+1],vx[basex][basey+1],distx,
  			     disty));
          }
      }
        
    	}
  
   
   
	if(j%factor<factor/2)
     {	
        
        basex = child_elec->locx;
    		basey = child_elec->locy+j/factor-1;
    		disty = 0.5 + (double)(j%factor)/(factor) + 0.5/(factor);
    
    	c_eys[0][j] = (1-tau)*interpolate1d(eys_old[basex][basey],
			      eys_old[basex][basey+1],disty) + 
			      (tau)*interpolate1d(eys[basex][basey],
			       eys[basex][basey+1],disty);
		  c_vy[0][j] = (1-tau)*interpolate1d(vy_old[basex][basey],
			     vy_old[basex][basey+1],disty) + 
			     (tau)*interpolate1d(vy[basex][basey],
			     vy[basex][basey+1],disty);
        
     }
	else
    {   
      
        basex = child_elec->locx;
    	  basey = child_elec->locy+j/factor;
	      disty = (double)(j%factor)/(factor) + 0.5/(factor)-0.5;

      	c_eys[0][j] = (1-tau)*interpolate1d(eys_old[basex][basey],
		      eys_old[basex][basey+1],disty) + (tau)*interpolate1d(
                      eys[basex][basey],eys[basex][basey+1],disty);
     	  c_vy[0][j] = (1-tau)*interpolate1d(vy_old[basex][basey],
		     vy_old[basex][basey+1],disty) + (tau)*interpolate1d(
		     vy[basex][basey],vy[basex][basey+1],disty);
              
        
    	
    }	
    
    if (j%factor<factor/2)
    {
      
		basex = child_elec->locx + (child_elec->m)/factor;
		basey = child_elec->locy+j/factor-1;
		disty = 0.5 + (double)(j%factor)/(factor) + 0.5/(factor);
		
  	c_eys[child_elec->m][j] = (1-tau)*(interpolate1d(eys_old[basex][basey],
  				  eys_old[basex][basey+1],disty))+(tau)*(
  				  interpolate1d(eys[basex][basey],
  				  eys[basex][basey+1],disty));
  
  	c_vy[child_elec->m][j] = (1-tau)*(interpolate1d(vy_old[basex][basey],
  				vy_old[basex][basey+1],disty))+(tau)*(
  				interpolate1d(vy[basex][basey],
  				vy[basex][basey+1],disty));
      
   }
  else
  {
  
     
      	basex = child_elec->locx + (child_elec->m)/factor;
        basey = child_elec->locy+j/factor;
      	disty = (double)(j%factor)/(factor) + 0.5/(factor)-0.5;

    		c_eys[child_elec->m][j] = (1-tau)*(interpolate1d(eys_old[basex][basey],
				eys_old[basex][basey+1],disty))+(tau)*(
				interpolate1d(eys[basex][basey],
				eys[basex][basey+1],disty));
	
      	c_vy[child_elec->m][j] = (1-tau)*(interpolate1d(vy_old[basex][basey],
      				vy_old[basex][basey+1],disty))+(tau)*(
      				interpolate1d(vy[basex][basey],
      				vy[basex][basey+1],disty));
       
      

  }

    if(j%factor<factor/2 && child_elec->locx>0)
    {
       
        	basex = child_elec->locx-1;
        	basey = child_elec->locy+j/factor-1;
        	distx = 0.5 + 0.5/factor;
        	disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
        	
          child_mag->mesh[0][j] = (1-tau)*(interpolate2d(hzi[basex][basey],
  				hzi[basex+1][basey],hzi[basex+1][basey+1],
  				hzi[basex][basey+1],distx,disty))+(tau)*(
  				interpolate2d(root_mag->mesh[basex][basey],
  				root_mag->mesh[basex+1][basey],
  				root_mag->mesh[basex+1][basey+1],
  				root_mag->mesh[basex][basey+1],distx,disty));
      
    }

    else if (child_elec->locx>0)
    {
     
      		basex = child_elec->locx-1;
      		basey = child_elec->locy+j/factor;
      		distx = 0.5 + 0.5/factor;
      		disty = 0.5/factor + (double)(j%factor)/factor-0.5;
      
      	child_mag->mesh[0][j] = (1-tau)*(interpolate2d(hzi[basex][basey],
				hzi[basex+1][basey],hzi[basex+1][basey+1],
				hzi[basex][basey+1],distx,disty))+(tau)*(
				interpolate2d(root_mag->mesh[basex][basey],
				root_mag->mesh[basex+1][basey],
				root_mag->mesh[basex+1][basey+1],
				root_mag->mesh[basex][basey+1],distx,disty));
     
    } 
	
   //!================ density ============
     
        	basex = child_elec->locx+i/factor;
        	basey = child_elec->locy+j/factor;
          distx = (double)(i%factor)/factor;
          disty = (double)(j%factor)/factor;
        
      
        	
        	child_den->mesh[i][j] = (1-tau)*(interpolate2d(denp[basex][basey],
        				denp[basex+1][basey],denp[basex+1][basey+1],
        				denp[basex][basey+1],distx,disty))+(tau)*(
        				interpolate2d(root_den->mesh[basex][basey],
        				root_den->mesh[basex+1][basey],
        				root_den->mesh[basex+1][basey+1],
        				root_den->mesh[basex][basey+1],distx,disty));
    
       
      //!===================================     
       
   
    
    }
}
    gettimeofday(&end,NULL);
    t_cal_interpolatex += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));
}

void interpolateyall(double t1)
{
     double tau = t1;
     int basex, basey;
     double distx,disty;
     //int factor=8;

for(j=0;j<=child_elec->n;j++ ) 
{
  for(i=1;i<child_elec->m;i++)
  {
	//int factor=2;
     double ita,ep;
  
    if((child_elec->locy>0) && (j<child_elec->n))
    {
     // if(j<(child_elec->n-1))
      if(j%factor<factor/2)
      {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor-1;
                distx = (double)(i%factor)/factor;
                //disty = 0.5 + 0.5/factor;
                disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],eys_old[basex][child_elec->locy],distx,disty))
                                                +(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))
                                                +(tau)*(interpolate2d(vy[basex][basey],vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      }
      
      else
      {
          if(child_elec->locy+j/factor+1<(root_elec->n))
          {
		            basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+j/factor;
                distx = (double)(i%factor)/factor;
                //disty = 0.5 + 0.5/factor;
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5;
                c_eys[i][j] = (1-tau)*(interpolate2d(eys_old[basex][basey],eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],eys_old[basex][child_elec->locy],distx,disty))
                                                +(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][j] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))
                                                +(tau)*(interpolate2d(vy[basex][basey],vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
      
         }
      }
      
      
    }
    
	if(i%factor<factor/2)
        {
         
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                c_exs[i][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
                                                
           
        }
	
	else
        {
           
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy;
                distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
                c_exs[i][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
          
        }
	
	if (i%factor<factor/2)
        {
          
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy+(child_elec->n)/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                c_exs[i][child_elec->n] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][child_elec->n] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
          
        }
	
	else
        {
          
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+(child_elec->n)/factor;
                distx = 0.5/factor + (double)(i%factor)/factor - 0.5;

                c_exs[i][child_elec->n] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][child_elec->n] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
          
        }
	
       	basex = child_elec->locx+(i)/factor;
        basey = child_elec->locy;
        distx = (double)(i%factor)/factor;
        child_den->mesh[i][0] = (1-tau)*(interpolate1d(denp[basex][basey],denp[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(root_den->mesh[basex][basey],root_den->mesh[basex+1][basey],distx));

        basex = child_elec->locx+(i)/factor;
        basey = child_elec->locy+(child_elec->n)/factor;
        distx = (double)(i%factor)/factor;
        child_den->mesh[i][child_elec->n] =(1-tau)*(interpolate1d(denp[basex][basey],denp[basex+1][basey],distx))
                                                                                +(tau)*(interpolate1d(root_den->mesh[basex][basey],root_den->mesh[basex+1][basey],distx));
 
 
   
       
          if((child_elec->locy>0) && (j<child_elec->n))
          {
            if(j%factor<factor/2)
            {
              
              if (i%factor<factor/2)
              {  
        
        
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy-1+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
               // disty = 0.5 + 0.5/factor;
               disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                                                                
              }
            
            else
            {
                
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor-1;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              //disty = 0.5 - 0.5/factor;
              disty = 0.5 + 0.5/factor + (double)(j%factor)/factor;
              child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))

               +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
                
             }  
            
            }    
                
       else    
        {
            if (i%factor<factor/2)
             {  
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy+j/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor; 
                disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
                child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            
              }
            else
            {
              basex = child_elec->locx+i/factor;
              basey = child_elec->locy+j/factor;
              distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
              //disty = 0.5 - 0.5/factor;
               disty = 0.5/factor+ (double)(j%factor)/factor - 0.5; 
               child_mag->mesh[i][j] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))

               +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
            }
        
          }        
      }              
                
                
          
         if(j<(child_elec->n))
         {
           	c_eyt[i][j] = c_eys[i][j]+c_eyi1[i][j];
            //c_eyt[i][child_elec->n-1] = c_eys[i][child_elec->n-1]+c_eyi1[i][child_elec->n-1];
         }
        
         if(j<=(child_elec->n))
         { 
            c_ext[i][j] = c_exs[i][j]+c_exi1[i][j];
          
         }
      
      
   }	
          

}



	// //========================================================================================================================================

}

void interpolatey(double t1)
{
     double tau = t1;
     int basex, basey;
     double distx,disty;
     
for(i=1;i<child_elec->m;i++)
{
	
    double ita,ep;
   if(child_elec->locy>0)
   {
		 basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy-1;
                distx = (double)(i%factor)/factor;
                disty = 0.5 + 0.5/factor;
                c_eys[i][0] = (1-tau)*(interpolate2d(eys_old[basex][basey],eys_old[basex+1][basey],eys_old[basex+1][child_elec->locy],eys_old[basex][child_elec->locy],distx,disty))
                                                +(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

                c_vy[i][0] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))
                                                +(tau)*(interpolate2d(vy[basex][basey],vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
   }
   
        basex = child_elec->locx+(i)/factor;
        basey = child_elec->locy+(child_elec->n-1)/factor;
        distx = (double)(i%factor)/factor;
        disty = 0.5 - 0.5/factor;
        c_eys[i][child_elec->n-1] = (1-tau)*(interpolate2d(eys_old[basex][basey],eys_old[basex+1][basey],eys_old[basex+1][basey+1],eys_old[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(eys[basex][basey],eys[basex+1][basey],eys[basex+1][basey+1],eys[basex][basey+1],distx,disty));

        c_vy[i][child_elec->n-1] = (1-tau)*(interpolate2d(vy_old[basex][basey],vy_old[basex+1][basey],vy_old[basex+1][basey+1],vy_old[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(vy[basex][basey],vy[basex+1][basey],vy[basex+1][basey+1],vy[basex][basey+1],distx,disty));
   
	if(i%factor<factor/2)
        {
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                c_exs[i][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
        }
	
	else
        {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy;
                distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
                c_exs[i][0] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][0] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
        }
	
	if (i%factor<factor/2)
        {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+(child_elec->n)/factor;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                c_exs[i][child_elec->n] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][child_elec->n] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
        }
	
	else
        {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy+(child_elec->n)/factor;
                distx = 0.5/factor + (double)(i%factor)/factor - 0.5;

                c_exs[i][child_elec->n] = (1-tau)*(interpolate1d(exs_old[basex][basey],exs_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(exs[basex][basey],exs[basex+1][basey],distx));

                c_vx[i][child_elec->n] = (1-tau)*(interpolate1d(vx_old[basex][basey],vx_old[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(vx[basex][basey],vx[basex+1][basey],distx));
        }
	
	basex = child_elec->locx+(i)/factor;
        basey = child_elec->locy;
        distx = (double)(i%factor)/factor;
        child_den->mesh[i][0] = (1-tau)*(interpolate1d(denp[basex][basey],denp[basex+1][basey],distx))
                                                        +(tau)*(interpolate1d(root_den->mesh[basex][basey],root_den->mesh[basex+1][basey],distx));

        basex = child_elec->locx+(i)/factor;
        basey = child_elec->locy+(child_elec->n)/factor;
        distx = (double)(i%factor)/factor;
        child_den->mesh[i][child_elec->n] =(1-tau)*(interpolate1d(denp[basex][basey],denp[basex+1][basey],distx))
                                                                                +(tau)*(interpolate1d(root_den->mesh[basex][basey],root_den->mesh[basex+1][basey],distx));
	
	if (i%factor<factor/2 && child_mag->locy>0)
        {
                basex = child_elec->locx+(i)/factor-1;
                basey = child_elec->locy-1;
                distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
                disty = 0.5 + 0.5/factor;
                child_mag->mesh[i][0] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
        }
	
	else if(child_mag->locy>0)
        {
                basex = child_elec->locx+(i)/factor;
                basey = child_elec->locy-1;
                distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
                disty = 0.5 + 0.5/factor;
                child_mag->mesh[i][0] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))
                                                                +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
        }
	
	if (i%factor<factor/2)
        {
                basex = child_elec->locx+i/factor-1;
        basey = child_elec->locy+(child_elec->n-1)/factor;
        distx = 0.5 + 0.5/factor + (double)(i%factor)/factor;
        disty = 0.5 - 0.5/factor;
                child_mag->mesh[i][child_elec->n-1] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))
                                                                                                +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
        }
	
	else
        {
                basex = child_elec->locx+i/factor;
        basey = child_elec->locy+(child_elec->n-1)/factor;
        distx = 0.5/factor + (double)(i%factor)/factor - 0.5;
        disty = 0.5 - 0.5/factor;
                child_mag->mesh[i][child_elec->n-1] = (1-tau)*(interpolate2d(hzi[basex][basey],hzi[basex+1][basey],hzi[basex+1][basey+1],hzi[basex][basey+1],distx,disty))

               +(tau)*(interpolate2d(root_mag->mesh[basex][basey],root_mag->mesh[basex+1][basey],root_mag->mesh[basex+1][basey+1],root_mag->mesh[basex][basey+1],distx,disty));
        }
	
	c_eyt[i][0] = c_eys[i][0]+c_eyi1[i][0];
    c_eyt[i][child_elec->n-1] = c_eys[i][child_elec->n-1]+c_eyi1[i][child_elec->n-1];
    c_ext[i][0] = c_exs[i][0]+c_exi1[i][0];
    c_ext[i][child_elec->n] = c_exs[i][child_elec->n]+c_exi1[i][child_elec->n];
	
}



	// //========================================================================================================================================

}

void c2p()
{	
    FILE *fpr;
    int l=1;
    int ic,jc;
    int cm1=0,cn1=0,cm2=0,cn2=0,cm3=0,cn3=0,cm4=0,cn4=0;
    int centr;
    gettimeofday(&begin,NULL);
    
    centr=(factor)/2;
    
     for(i=factor;i<child_elec->m-factor;i+=factor)
        {
        
              
                for(j=factor;j<child_elec->n;j+=factor)
                {
                   
                   
                   exs[child_elec->locx+i/factor][child_elec->locy+j/factor]
                 = (c_exs[i+centr-1][j]+c_exs[i+centr][j])/2;
                 
                 vx[child_elec->locx+i/factor][child_elec->locy+j/factor]
                 = (c_vx[i+centr-1][j]+c_vx[i+centr][j])/2;
                
                ext[child_elec->locx+i/factor][child_elec->locy+j/factor]
                 = exs[child_elec->locx+i/factor][child_elec->locy+j/factor]
                + exi1[child_elec->locx+i/factor][child_elec->locy+j/factor];
                 

                }
    
              
        }
        
	
	for(i=factor;i<child_elec->m;i+=factor)
        {
            
                for(j=factor;j<child_elec->n-(factor);j+=(factor))
                {    
                    
                    
                   eys[child_elec->locx+i/factor][child_elec->locy+j/factor]
                   = (c_eys[i][j+centr-1]+c_eys[i][j+centr])/2;
                   
                  eyt[child_elec->locx+i/factor][child_elec->locy+j/factor]
                   = eys[child_elec->locx+i/factor][child_elec->locy+j/factor]
                   + eyi1[child_elec->locx+i/factor][child_elec->locy+j/factor];
               
                   vy[child_elec->locx+i/factor][child_elec->locy+j/factor]
                   = (c_vy[i][j+centr-1]+c_vy[i][j+centr])/2;
                 
                    
                 }
                 
        }
	
	for(i=factor;i<child_elec->m;i+=factor)
        {
                for(j=factor;j<child_elec->n;j+=factor)
                {
                
                                
                 
                 root_den->mesh[child_den->locx+(i)/factor][child_den->locy+(j)/factor]
              = child_den->mesh[i][j];
                
                   
            
            
                }
        }
	
	for(i=factor;i<child_elec->m-(factor);i+=(factor))
        {
              
        
                for(j=factor;j<child_elec->n-(factor);j+=(factor))
                {
                    
              
		
		root_mag->mesh[child_mag->locx+i/factor][child_mag->locy+j/factor]
              = (child_mag->mesh[i+centr-1][j+centr-1]
                +child_mag->mesh[i+centr][j+centr-1]
                +child_mag->mesh[i+centr-1][j+centr]
                +child_mag->mesh[i+centr][j+centr])/4;
                    
                   
                }
                
        } 
	 
    gettimeofday(&end,NULL);
    t_cal_c2p += ((end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0));

}



